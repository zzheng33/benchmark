#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

#include <omp.h>

#include <hip/hip_runtime.h>

#define THREADS_PER_DIM 16
#define BLOCKS_PER_DIM 16
#define THREADS_PER_BLOCK THREADS_PER_DIM*THREADS_PER_DIM

#include "kmeans_cuda_kernel.cu"


//#define BLOCK_DELTA_REDUCE
//#define BLOCK_CENTER_REDUCE

#define CPU_DELTA_REDUCE
#define CPU_CENTER_REDUCE

extern "C"
int setup(int argc, char** argv);									/* function prototype */

// GLOBAL!!!!!
unsigned int num_threads_perdim = THREADS_PER_DIM;					/* sqrt(256) -- see references for this choice */
unsigned int num_blocks_perdim = BLOCKS_PER_DIM;					/* temporary */
unsigned int num_threads = num_threads_perdim*num_threads_perdim;	/* number of threads */
unsigned int num_blocks = num_blocks_perdim*num_blocks_perdim;		/* number of blocks */

/* _d denotes it resides on the device */
int    *membership_new;												/* newly assignment membership */
float  *feature_d;													/* inverted data array */
float  *feature_flipped_d;											/* original (not inverted) data array */
int    *membership_d;												/* membership on the device */
float  *block_new_centers;											/* sum of points in a cluster (per block) */
float  *clusters_d;													/* cluster centers on the device */
float  *block_clusters_d;											/* per block calculation of cluster centers */
int    *block_deltas_d;												/* per block calculation of deltas */


/* -------------- allocateMemory() ------------------- */
/* allocate device memory, calculate number of blocks and threads, and invert the data array */
extern "C"
void allocateMemory(int npoints, int nfeatures, int nclusters, float **features)
{	
	num_blocks = npoints / num_threads;
	if (npoints % num_threads > 0)		/* defeat truncation */
		num_blocks++;

	num_blocks_perdim = sqrt((double) num_blocks);
	while (num_blocks_perdim * num_blocks_perdim < num_blocks)	// defeat truncation (should run once)
		num_blocks_perdim++;

	num_blocks = num_blocks_perdim*num_blocks_perdim;

	/* allocate memory for memory_new[] and initialize to -1 (host) */
	membership_new = (int*) malloc(npoints * sizeof(int));
	for(int i=0;i<npoints;i++) {
		membership_new[i] = -1;
	}

	/* allocate memory for block_new_centers[] (host) */
	block_new_centers = (float *) malloc(nclusters*nfeatures*sizeof(float));
	
	/* allocate memory for feature_flipped_d[][], feature_d[][] (device) */
	hipMalloc((void**) &feature_flipped_d, npoints*nfeatures*sizeof(float));
	hipMemcpy(feature_flipped_d, features[0], npoints*nfeatures*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**) &feature_d, npoints*nfeatures*sizeof(float));
		
	/* invert the data array (kernel execution) */	
	invert_mapping<<<num_blocks,num_threads>>>(feature_flipped_d,feature_d,npoints,nfeatures);
		
	/* allocate memory for membership_d[] and clusters_d[][] (device) */
	hipMalloc((void**) &membership_d, npoints*sizeof(int));
	hipMalloc((void**) &clusters_d, nclusters*nfeatures*sizeof(float));

	
#ifdef BLOCK_DELTA_REDUCE
	// allocate array to hold the per block deltas on the gpu side
	
	hipMalloc((void**) &block_deltas_d, num_blocks_perdim * num_blocks_perdim * sizeof(int));
	//hipMemcpy(block_delta_d, &delta_h, sizeof(int), hipMemcpyHostToDevice);
#endif

#ifdef BLOCK_CENTER_REDUCE
	// allocate memory and copy to card cluster  array in which to accumulate center points for the next iteration
	hipMalloc((void**) &block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float));
	//hipMemcpy(new_clusters_d, new_centers[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice);
#endif

}
/* -------------- allocateMemory() end ------------------- */

/* -------------- deallocateMemory() ------------------- */
/* free host and device memory */
extern "C"
void deallocateMemory()
{
	free(membership_new);
	free(block_new_centers);
	hipFree(feature_d);
	hipFree(feature_flipped_d);
	hipFree(membership_d);

	hipFree(clusters_d);
#ifdef BLOCK_CENTER_REDUCE
    hipFree(block_clusters_d);
#endif
#ifdef BLOCK_DELTA_REDUCE
    hipFree(block_deltas_d);
#endif
}
/* -------------- deallocateMemory() end ------------------- */



////////////////////////////////////////////////////////////////////////////////
// Program main																  //

int
main( int argc, char** argv) 
{
	// make sure we're running on the big card
    hipSetDevice(1);
	// as done in the CUDA start/help document provided
	setup(argc, argv);    
}

//																			  //
////////////////////////////////////////////////////////////////////////////////


/* ------------------- kmeansCuda() ------------------------ */    
extern "C"
int kmeansCuda(float  **feature,                /* in: [npoints][nfeatures] */
               int      nfeatures,              /* number of attributes for each point */
               int      npoints,                /* number of data points */
               int      nclusters,              /* number of clusters */
               int     *membership,             /* which cluster the point belongs to */
               float  **clusters,               /* coordinates of cluster centers */
               int     *new_centers_len,        /* number of elements in each cluster */
               float  **new_centers             /* sum of elements in each cluster */
              )
{
    int delta = 0;            /* if point has moved */
    int i, j;                 /* counters */

    hipSetDevice(1);

    /* copy membership (host to device) */
    hipMemcpy(membership_d, membership_new, npoints * sizeof(int), hipMemcpyHostToDevice);

    /* copy clusters (host to device) */
    hipMemcpy(clusters_d, clusters[0], nclusters * nfeatures * sizeof(float), hipMemcpyHostToDevice);

    // Create texture objects
    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    memset(&texDesc, 0, sizeof(texDesc));
    
    resDesc.resType = hipResourceTypeLinear;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object for t_features
    resDesc.res.linear.devPtr = feature_d;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.sizeInBytes = npoints * nfeatures * sizeof(float);
    hipCreateTextureObject(&t_features, &resDesc, &texDesc, NULL);

    // Create texture object for t_features_flipped
    resDesc.res.linear.devPtr = feature_flipped_d;
    hipCreateTextureObject(&t_features_flipped, &resDesc, &texDesc, NULL);

    // Create texture object for t_clusters
    resDesc.res.linear.devPtr = clusters_d;
    resDesc.res.linear.sizeInBytes = nclusters * nfeatures * sizeof(float);
    hipCreateTextureObject(&t_clusters, &resDesc, &texDesc, NULL);

    /* copy clusters to constant memory */
    hipMemcpyToSymbol(HIP_SYMBOL(c_clusters), clusters[0], nclusters * nfeatures * sizeof(float), 0, hipMemcpyHostToDevice);

    /* setup execution parameters */
    dim3 grid(num_blocks_perdim, num_blocks_perdim);
    dim3 threads(num_threads_perdim * num_threads_perdim);

    /* execute the kernel */
   kmeansPoint<<<grid, threads>>>(t_features, t_clusters, nfeatures, npoints, nclusters, membership_d, clusters_d, block_clusters_d, block_deltas_d);


    hipDeviceSynchronize();

    /* copy back membership (device to host) */
    hipMemcpy(membership_new, membership_d, npoints * sizeof(int), hipMemcpyDeviceToHost);

    /* destroy texture objects */
    hipDestroyTextureObject(t_features);
    hipDestroyTextureObject(t_features_flipped);
    hipDestroyTextureObject(t_clusters);

    /* for each point, sum data points in each cluster and see if membership has changed */
    delta = 0;
    for (i = 0; i < npoints; i++)
    {        
        int cluster_id = membership_new[i];
        new_centers_len[cluster_id]++;
        if (membership_new[i] != membership[i])
        {
            delta++;
            membership[i] = membership_new[i];
        }
        for (j = 0; j < nfeatures; j++)
        {            
            new_centers[cluster_id][j] += feature[i][j];
        }
    }

    return delta;
}

/* ------------------- kmeansCuda() end ------------------------ */    

