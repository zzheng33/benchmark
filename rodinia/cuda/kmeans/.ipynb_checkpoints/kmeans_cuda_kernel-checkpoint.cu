#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kmeans.h"

#define ASSUMED_NR_CLUSTERS 32

__constant__ float c_clusters[ASSUMED_NR_CLUSTERS * 34]; /* constant memory for cluster centers */

hipTextureObject_t t_features, t_features_flipped, t_clusters;

/* Function to create texture objects */
void createTextureObjects(float *d_features, float *d_features_flipped, float *d_clusters, size_t npoints, size_t nfeatures, size_t nclusters) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Texture for features
    resDesc.res.linear.devPtr = d_features;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.sizeInBytes = npoints * nfeatures * sizeof(float);
    hipCreateTextureObject(&t_features, &resDesc, &texDesc, NULL);

    // Texture for flipped features
    resDesc.res.linear.devPtr = d_features_flipped;
    resDesc.res.linear.sizeInBytes = npoints * nfeatures * sizeof(float);
    hipCreateTextureObject(&t_features_flipped, &resDesc, &texDesc, NULL);

    // Texture for clusters
    resDesc.res.linear.devPtr = d_clusters;
    resDesc.res.linear.sizeInBytes = nclusters * nfeatures * sizeof(float);
    hipCreateTextureObject(&t_clusters, &resDesc, &texDesc, NULL);
}

/* Function to destroy texture objects */
void destroyTextureObjects() {
    hipDestroyTextureObject(t_features);
    hipDestroyTextureObject(t_features_flipped);
    hipDestroyTextureObject(t_clusters);
}

/* ----------------- invert_mapping() --------------------- */
__global__ void invert_mapping(float *input, float *output, int npoints, int nfeatures) {
    int point_id = threadIdx.x + blockDim.x * blockIdx.x;
    int i;

    if (point_id < npoints) {
        for (i = 0; i < nfeatures; i++)
            output[point_id + npoints * i] = input[point_id * nfeatures + i];
    }
}

/* ----------------- kmeansPoint() --------------------- */
__global__ void kmeansPoint(hipTextureObject_t t_features, 
                            hipTextureObject_t t_clusters, 
                            int nfeatures, 
                            int npoints, 
                            int nclusters, 
                            int *membership, 
                            float *clusters, 
                            float *block_clusters, 
                            int *block_deltas) 
{
    const unsigned int block_id = gridDim.x * blockIdx.y + blockIdx.x;
    const unsigned int point_id = block_id * blockDim.x * blockDim.y + threadIdx.x;

    int index = -1;

    if (point_id < npoints) {
        int i, j;
        float min_dist = FLT_MAX;
        float dist;

        for (i = 0; i < nclusters; i++) {
            int cluster_base_index = i * nfeatures;
            float ans = 0.0;

            for (j = 0; j < nfeatures; j++) {
                int addr = point_id + j * npoints;
                float diff = tex1D<float>(t_features, addr) - c_clusters[cluster_base_index + j];
                ans += diff * diff;
            }
            dist = ans;

            if (dist < min_dist) {
                min_dist = dist;
                index = i;
            }
        }
    }

#ifdef GPU_DELTA_REDUCTION
    __shared__ int deltas[THREADS_PER_BLOCK];
    if (threadIdx.x < THREADS_PER_BLOCK) {
        deltas[threadIdx.x] = 0;
    }
#endif

    if (point_id < npoints) {
#ifdef GPU_DELTA_REDUCTION
        if (membership[point_id] != index) {
            deltas[threadIdx.x] = 1;
        }
#endif
        membership[point_id] = index;
    }

#ifdef GPU_DELTA_REDUCTION
    __syncthreads();
    unsigned int threadids_participating = THREADS_PER_BLOCK / 2;
    for (; threadids_participating > 1; threadids_participating /= 2) {
        if (threadIdx.x < threadids_participating) {
            deltas[threadIdx.x] += deltas[threadIdx.x + threadids_participating];
        }
        __syncthreads();
    }
    if (threadIdx.x < 1) {
        deltas[threadIdx.x] += deltas[threadIdx.x + 1];
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        block_deltas[blockIdx.y * gridDim.x + blockIdx.x] = deltas[0];
    }
#endif

#ifdef GPU_NEW_CENTER_REDUCTION
    int center_id = threadIdx.x / nfeatures;
    int dim_id = threadIdx.x - nfeatures * center_id;

    __shared__ int new_center_ids[THREADS_PER_BLOCK];

    new_center_ids[threadIdx.x] = index;
    __syncthreads();

    int new_base_index = (point_id - threadIdx.x) * nfeatures + dim_id;
    float accumulator = 0.f;

    if (threadIdx.x < nfeatures * nclusters) {
        for (int i = 0; i < THREADS_PER_BLOCK; i++) {
            float val = tex1D<float>(t_features_flipped, new_base_index + i * nfeatures);
            if (new_center_ids[i] == center_id)
                accumulator += val;
        }

        block_clusters[(blockIdx.y * gridDim.x + blockIdx.x) * nclusters * nfeatures + threadIdx.x] = accumulator;
    }
#endif
}

#endif // #ifndef _KMEANS_CUDA_KERNEL_H_

