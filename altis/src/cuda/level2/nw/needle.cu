////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\level2\nw\needle.cu
//
// summary:	Needle class
// 
// origin: Rodinia (http://rodinia.cs.virginia.edu/doku.php)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <limits.h>

#include "cudacommon.h"
#include "OptionParser.h"
#include "ResultDatabase.h"
#include "needle.h"
#include "needle_kernel.cu"

#define LIMIT -999
#define SEED 7

int max_rows, max_cols, penalty;

void runTest(ResultDatabase &resultDB, OptionParser &op);

/// <summary>	The blosum 62[24][24]. </summary>
int blosum62[24][24] = {{4,  -1, -2, -2, 0, -1, -1, 0, -2, -1, -1, -1,
                         -1, -2, -1, 1,  0, -3, -2, 0, -2, -1, 0,  -4},
                        {-1, 5,  0,  -2, -3, 1,  0,  -2, 0,  -3, -2, 2,
                         -1, -3, -2, -1, -1, -3, -2, -3, -1, 0,  -1, -4},
                        {-2, 0,  6,  1, -3, 0,  0,  0,  1, -3, -3, 0,
                         -2, -3, -2, 1, 0,  -4, -2, -3, 3, 0,  -1, -4},
                        {-2, -2, 1,  6, -3, 0,  2,  -1, -1, -3, -4, -1,
                         -3, -3, -1, 0, -1, -4, -3, -3, 4,  1,  -1, -4},
                        {0,  -3, -3, -3, 9,  -3, -4, -3, -3, -1, -1, -3,
                         -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
                        {-1, 1,  0,  0, -3, 5,  2,  -2, 0, -3, -2, 1,
                         0,  -3, -1, 0, -1, -2, -1, -2, 0, 3,  -1, -4},
                        {-1, 0,  0,  2, -4, 2,  5,  -2, 0, -3, -3, 1,
                         -2, -3, -1, 0, -1, -3, -2, -2, 1, 4,  -1, -4},
                        {0,  -2, 0,  -1, -3, -2, -2, 6,  -2, -4, -4, -2,
                         -3, -3, -2, 0,  -2, -2, -3, -3, -1, -2, -1, -4},
                        {-2, 0,  1,  -1, -3, 0,  0, -2, 8, -3, -3, -1,
                         -2, -1, -2, -1, -2, -2, 2, -3, 0, 0,  -1, -4},
                        {-1, -3, -3, -3, -1, -3, -3, -4, -3, 4,  2,  -3,
                         1,  0,  -3, -2, -1, -3, -1, 3,  -3, -3, -1, -4},
                        {-1, -2, -3, -4, -1, -2, -3, -4, -3, 2,  4,  -2,
                         2,  0,  -3, -2, -1, -2, -1, 1,  -4, -3, -1, -4},
                        {-1, 2,  0,  -1, -3, 1,  1,  -2, -1, -3, -2, 5,
                         -1, -3, -1, 0,  -1, -3, -2, -2, 0,  1,  -1, -4},
                        {-1, -1, -2, -3, -1, 0,  -2, -3, -2, 1,  2,  -1,
                         5,  0,  -2, -1, -1, -1, -1, 1,  -3, -1, -1, -4},
                        {-2, -3, -3, -3, -2, -3, -3, -3, -1, 0,  0,  -3,
                         0,  6,  -4, -2, -2, 1,  3,  -1, -3, -3, -1, -4},
                        {-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1,
                         -2, -4, 7,  -1, -1, -4, -3, -2, -2, -1, -2, -4},
                        {1,  -1, 1,  0, -1, 0,  0,  0,  -1, -2, -2, 0,
                         -1, -2, -1, 4, 1,  -3, -2, -2, 0,  0,  0,  -4},
                        {0,  -1, 0,  -1, -1, -1, -1, -2, -2, -1, -1, -1,
                         -1, -2, -1, 1,  5,  -2, -2, 0,  -1, -1, 0,  -4},
                        {-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3,
                         -1, 1,  -4, -3, -2, 11, 2,  -3, -4, -3, -2, -4},
                        {-2, -2, -2, -3, -2, -1, -2, -3, 2,  -1, -1, -2,
                         -1, 3,  -3, -2, -2, 2,  7,  -1, -3, -2, -1, -4},
                        {0, -3, -3, -3, -1, -2, -2, -3, -3, 3,  1,  -2,
                         1, -1, -2, -2, 0,  -3, -1, 4,  -3, -2, -1, -4},
                        {-2, -1, 3,  4, -3, 0,  1,  -1, 0, -3, -4, 0,
                         -3, -3, -2, 0, -1, -4, -3, -3, 4, 1,  -1, -4},
                        {-1, 0,  0,  1, -3, 3,  4,  -2, 0, -3, -3, 1,
                         -1, -3, -1, 0, -1, -3, -2, -2, 1, 4,  -1, -4},
                        {0,  -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1,
                         -1, -1, -2, 0,  0,  -2, -1, -1, -1, -1, -1, -4},
                        {-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,
                         -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, 1}};


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Add benchmark specific options parsing.  The user is allowed to specify
/// the size of the input data in kiB.. </summary>
///
/// <remarks>	Ed, 5/20/2020.
/// 			Anthony Danalis, 9/08, 2009
///
/// <param name="op">	[in,out] the options parser / parameter database. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void addBenchmarkSpecOptions(OptionParser &op) {
  op.addOption("uvm", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory, only demand paging");
  op.addOption("dimensions", OPT_INT, "0", "dimensions");
  op.addOption("penalty", OPT_INT, "10", "penalty");
  op.addOption("resultsfile", OPT_STRING, "", "file to write results to");
}


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the benchmark operation. </summary>
///
/// <remarks>	Ed, 5/20/2020.
/// 			Kyle Spafford, 8/13/2009 </remarks>
///
/// <param name="resultDB">	[in,out] results from the benchmark are stored in this db. </param>
/// <param name="op">	   	[in,out] the options parser / parameter database. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
  printf("Running Needleman-Wunsch\n");

  int device;
  hipGetDevice(&device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

  bool quiet = op.getOptionBool("quiet");
  int dim = op.getOptionInt("dimensions");
  penalty = op.getOptionInt("penalty");

  if(penalty < 0) {
      printf("Error: Penalty must be a positive number.\n");
      return;
  }
  if(dim < 0) {
      printf("Error: Dimensions must be positive.\n");
      return;
  }

  if (dim == 0) {
    int problemSizes[4] = {100, 1000, 6000, 40000};
    dim = problemSizes[op.getOptionInt("size") - 1];
  }

  long long num_items = (long long)dim * (long long)dim;
  if(num_items >= INT_MAX) {
      printf("Error: Total size cannot exceed INT_MAX");
      return;
  }

  if(!quiet) {
      printf("WG size of kernel = %d \n", BLOCK_SIZE);
      printf("Max Rows x Cols: %dx%d\n", dim, dim);
      printf("Penalty: %d\n\n", penalty);
  }
  srand(SEED);

  int passes = op.getOptionInt("passes");
  for (int i = 0; i < passes; i++) {
      if(!quiet) {
          printf("Pass %d: ", i);
      }
      max_rows = dim;
      max_cols = dim;
      runTest(resultDB, op);
      if(!quiet) {
          printf("Done.\n");
      }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the test operation. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="op">	   	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void runTest(ResultDatabase &resultDB, OptionParser &op) {
  bool uvm = op.getOptionBool("uvm");
  bool quiet = op.getOptionBool("quiet");
  int *input_itemsets, *output_itemsets, *referrence;
  int *matrix_cuda, *referrence_cuda;
  int size;

  max_rows = max_rows + 1;
  max_cols = max_cols + 1;
  
  if (uvm) {
    checkCudaErrors(hipMallocManaged(&referrence, max_rows * max_cols * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&input_itemsets, max_rows * max_cols * sizeof(int)));
  } else {
    referrence = (int *)malloc(max_rows * max_cols * sizeof(int));
    assert(referrence);
    input_itemsets = (int *)malloc(max_rows * max_cols * sizeof(int));
    assert(input_itemsets);
    output_itemsets = (int *)malloc(max_rows * max_cols * sizeof(int));
    assert(output_itemsets);
  }

  if (!input_itemsets) {
      fprintf(stderr, "Error: Can not allocate memory\n");
      exit(0);
  }

  for (int i = 0; i < max_cols; i++) {
    for (int j = 0; j < max_rows; j++) {
      input_itemsets[i * max_cols + j] = 0;
    }
  }

  for (int i = 1; i < max_rows; i++) {  // please define your own sequence.
    input_itemsets[i * max_cols] = rand() % 10 + 1;
  }
  for (int j = 1; j < max_cols; j++) {  // please define your own sequence.
    input_itemsets[j] = rand() % 10 + 1;
  }

  for (int i = 1; i < max_cols; i++) {
    for (int j = 1; j < max_rows; j++) {
      referrence[i * max_cols + j] =
          blosum62[input_itemsets[i * max_cols]][input_itemsets[j]];
    }
  }

  for (int i = 1; i < max_rows; i++)
    input_itemsets[i * max_cols] = -i * penalty;
  for (int j = 1; j < max_cols; j++) input_itemsets[j] = -j * penalty;

  size = max_cols * max_rows;

  if (uvm) {
    // Do nothing
  } else {
    checkCudaErrors(hipMalloc((void **)&referrence_cuda, sizeof(int) * size));
    checkCudaErrors(hipMalloc((void **)&matrix_cuda, sizeof(int) * size));
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;
  double transferTime = 0.;
  double kernelTime = 0;

  hipEventRecord(start, 0);
  // Notice that here we used demand paging so no cpy time included, could also use HyperQ
  if (uvm) {
    referrence_cuda = referrence;
    matrix_cuda = input_itemsets;
  } else {
    checkCudaErrors(hipMemcpy(referrence_cuda, referrence, sizeof(int) * size,
            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(matrix_cuda, input_itemsets, sizeof(int) * size,
            hipMemcpyHostToDevice));
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds

  dim3 dimGrid;
  dim3 dimBlock(BLOCK_SIZE, 1);
  int block_width = (max_cols - 1) / BLOCK_SIZE;

  // process top-left matrix
  for (int i = 1; i <= block_width; i++) {
    dimGrid.x = i;
    dimGrid.y = 1;
    hipEventRecord(start, 0);
    needle_cuda_shared_1<<<dimGrid, dimBlock>>>(
            referrence_cuda, matrix_cuda, max_cols, penalty, i, block_width);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    kernelTime += elapsedTime * 1.e-3;
    CHECK_CUDA_ERROR();
  }
  // process bottom-right matrix
  for (int i = block_width - 1; i >= 1; i--) {
    dimGrid.x = i;
    dimGrid.y = 1;
    hipEventRecord(start, 0);
    needle_cuda_shared_2<<<dimGrid, dimBlock>>>(
        referrence_cuda, matrix_cuda, max_cols, penalty, i, block_width);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    kernelTime += elapsedTime * 1.e-3;
    CHECK_CUDA_ERROR();
  }

  hipEventRecord(start, 0);
  if (uvm) {
    output_itemsets = matrix_cuda;
    checkCudaErrors(hipMemPrefetchAsync(output_itemsets, sizeof(int) * size, hipCpuDeviceId));
    checkCudaErrors(hipStreamSynchronize(0));
  } else {
    checkCudaErrors(hipMemcpy(output_itemsets, matrix_cuda, sizeof(int) * size,
            hipMemcpyDeviceToHost));
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds

  string outfile = op.getOptionString("outputFile");
  if (outfile != "") {
      FILE *fpo = fopen(outfile.c_str(), "w");
      if(!quiet) {
        fprintf(fpo, "Print traceback value GPU to %s:\n", outfile.c_str());
      }

      for (int i = max_rows - 2, j = max_rows - 2; i >= 0, j >= 0;) {
          int nw, n, w, traceback;
          if (i == max_rows - 2 && j == max_rows - 2) {
              // print the first element
              fprintf(fpo, "%d ", output_itemsets[i*max_cols+j]);
          }
          if (i == 0 && j == 0) {
              break;
          }
          if (i > 0 && j > 0) {
              nw = output_itemsets[(i - 1) * max_cols + j - 1];
              w = output_itemsets[i * max_cols + j - 1];
              n = output_itemsets[(i - 1) * max_cols + j];
          } else if (i == 0) {
              nw = n = LIMIT;
              w = output_itemsets[i * max_cols + j - 1];
          } else if (j == 0) {
              nw = w = LIMIT;
              n = output_itemsets[(i - 1) * max_cols + j];
          } else {
          }

          // traceback = maximum(nw, w, n);
          int new_nw, new_w, new_n;
          new_nw = nw + referrence[i * max_cols + j];
          new_w = w - penalty;
          new_n = n - penalty;

          traceback = maximum(new_nw, new_w, new_n);
          if (traceback == new_nw) {
              traceback = nw;
          }
          if (traceback == new_w) {
              traceback = w;
          }
          if (traceback == new_n) {
              traceback = n;
          }

          fprintf(fpo, "%d ", traceback);
          if (traceback == nw) {
              i--;
              j--;
              continue;
          } else if (traceback == w) {
              j--;
              continue;
          } else if (traceback == n) {
              i--;
              continue;
          } else {
          }
      }
      fclose(fpo);
  }

  // Cleanup memory
  if (uvm) {
    checkCudaErrors(hipFree(referrence_cuda));
    checkCudaErrors(hipFree(matrix_cuda));
  } else {
    checkCudaErrors(hipFree(referrence_cuda));
    checkCudaErrors(hipFree(matrix_cuda));
    free(referrence);
    free(input_itemsets);
    free(output_itemsets);
  }

  char tmp[32];
  sprintf(tmp, "%ditems", size);
  string atts = string(tmp);
  resultDB.AddResult("NW-TransferTime", atts, "sec", transferTime);
  resultDB.AddResult("NW-KernelTime", atts, "sec", kernelTime);
  resultDB.AddResult("NW-TotalTime", atts, "sec", transferTime + kernelTime);
  resultDB.AddResult("NW-Rate_Parity", atts, "N", transferTime / kernelTime);
  resultDB.AddOverall("Time", "sec", kernelTime+transferTime);
}
