#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Modfified by Bodun Hu <bodunhu@utexas.edu>
 * Added: UVM and coop support
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "polybenchUtilFuncts.h"

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float, int, and double */
typedef float DATA_TYPE;

struct fdtd_params {
    int NX;
    int NY;
    DATA_TYPE *_fict_;
    DATA_TYPE *ex;
    DATA_TYPE *ey;
    DATA_TYPE *hz;
    int t;
};


void init_arrays(size_t NX, size_t NY, size_t tmax, DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
    int i, j;

    for (i = 0; i < tmax; i++)
    {
        _fict_[i] = (DATA_TYPE) i;
    }
    
    for (i = 0; i < NX; i++)
    {
        for (j = 0; j < NY; j++)
        {
            ex[i*NY + j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
            ey[i*NY + j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
            hz[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
        }
    }
}


void runFdtd(size_t NX, size_t NY, size_t tmax, DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
    int t, i, j;
    
    for (t=0; t < tmax; t++)  
    {
        for (j=0; j < NY; j++)
        {
            ey[0*NY + j] = _fict_[t];
        }
    
        for (i = 1; i < NX; i++)
        {
            for (j = 0; j < NY; j++)
            {
                ey[i*NY + j] = ey[i*NY + j] - 0.5*(hz[i*NY + j] - hz[(i-1)*NY + j]);
            }
        }

        for (i = 0; i < NX; i++)
        {
            for (j = 1; j < NY; j++)
            {
                ex[i*(NY+1) + j] = ex[i*(NY+1) + j] - 0.5*(hz[i*NY + j] - hz[i*NY + (j-1)]);
            }
        }

        for (i = 0; i < NX; i++)
        {
            for (j = 0; j < NY; j++)
            {
                hz[i*NY + j] = hz[i*NY + j] - 0.7*(ex[i*(NY+1) + (j+1)] - ex[i*(NY+1) + j] + ey[(i+1)*NY + j] - ey[i*NY + j]);
            }
        }
    }
}


void compareResults(size_t NX, size_t NY, DATA_TYPE* hz1, DATA_TYPE* hz2)
{
    int i, j, fail;
    fail = 0;
    
    for (i=0; i < NX; i++) 
    {
        for (j=0; j < NY; j++) 
        {
            if (percentDiff(hz1[i*NY + j], hz2[i*NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
            {
                fail++;
            }
        }
    }
    
    // Print results
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

__global__ void fdtd_step1_kernel(size_t NX, size_t NY, DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < NX) && (j < NY))
    {
        if (i == 0) 
        {
            ey[i * NY + j] = _fict_[t];
        }
        else
        { 
            ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
        }
    }
}



__global__ void fdtd_step2_kernel(size_t NX, size_t NY, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    if ((i < NX) && (j < NY) && (j > 0))
    {
        ex[i * (NY+1) + j] = ex[i * (NY+1) + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
    }
}


__global__ void fdtd_step3_kernel(size_t NX, size_t NY, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    if ((i < NX) && (j < NY))
    {	
        hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * (NY+1) + (j+1)] - ex[i * (NY+1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
    }
}

__global__ void fdtd_coop_kernel(fdtd_params params)
{
    int NX = params.NX;
    int NY = params.NY;
    DATA_TYPE *_fict_ = params._fict_;
    DATA_TYPE *ex = params.ex;
    DATA_TYPE *ey = params.ey;
    DATA_TYPE *hz = params.hz;
    int t = params.t;
    
    grid_group g = this_grid();
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // kernel 1
    if ((i < NX) && (j < NY))
    {
        if (i == 0) 
        {
            ey[i * NY + j] = _fict_[t];
        }
        else
        { 
            ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
        }
    }

    // kernel 2
    if ((i < NX) && (j < NY) && (j > 0))
    {
        ex[i * (NY+1) + j] = ex[i * (NY+1) + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
    }

    g.sync();

    // kernel 3
    if ((i < NX) && (j < NY))
    {
        hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * (NY+1) + (j+1)] - ex[i * (NY+1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
    }
}


void fdtdCuda(size_t NX, size_t NY, size_t tmax, DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz, DATA_TYPE* hz_outputFromGpu,
            ResultDatabase &DB, OptionParser &op)
{
    const bool uvm = op.getOptionBool("uvm");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));

    double t_start, t_end;

    DATA_TYPE *_fict_gpu;
    DATA_TYPE *ex_gpu;
    DATA_TYPE *ey_gpu;
    DATA_TYPE *hz_gpu;


    checkCudaErrors(hipMalloc((void **)&_fict_gpu, sizeof(DATA_TYPE) * tmax));
    checkCudaErrors(hipMalloc((void **)&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1)));
    checkCudaErrors(hipMalloc((void **)&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY));
    checkCudaErrors(hipMalloc((void **)&hz_gpu, sizeof(DATA_TYPE) * NX * NY));

    checkCudaErrors(hipMemcpy(_fict_gpu, _fict_, sizeof(DATA_TYPE) * tmax, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ex_gpu, ex, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ey_gpu, ey, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(hz_gpu, hz, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice));

    dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
    dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)), (size_t)ceil(((float)NX) / ((float)block.y)));

    t_start = rtclock();

    if (op.getOptionBool("coop"))
    {
        fdtd_params params;
        params.NX = NX;
        params.NY = NY;
        params._fict_ = _fict_gpu;
        params.ex = ex_gpu;
        params.ey = ey_gpu;
        params.hz = hz_gpu;
        void *p_params = {&params};
        for (int t = 0; t < tmax; t++)
        {
            params.t = t;
            checkCudaErrors(hipLaunchCooperativeKernel((void*)fdtd_coop_kernel, grid, block, &p_params));
        }
    }
    else
    {
        hipStream_t stream1, stream2;
        checkCudaErrors(hipStreamCreate(&stream1));
        checkCudaErrors(hipStreamCreate(&stream2));
        for (int t = 0; t < tmax; t++)
        {
            fdtd_step1_kernel<<<grid,block,0,stream1>>>(NX, NY, _fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
            fdtd_step2_kernel<<<grid,block,0,stream2>>>(NX, NY, ex_gpu, ey_gpu, hz_gpu, t);
            fdtd_step3_kernel<<<grid,block>>>(NX, NY, ex_gpu, ey_gpu, hz_gpu, t);
        }
        checkCudaErrors(hipStreamDestroy(stream1));
        checkCudaErrors(hipStreamDestroy(stream2));
    }
    hipDeviceSynchronize();
    t_end = rtclock();
    fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

    checkCudaErrors(hipMemcpy(hz_outputFromGpu, hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(_fict_gpu));
    checkCudaErrors(hipFree(ex_gpu));
    checkCudaErrors(hipFree(ey_gpu));
    checkCudaErrors(hipFree(hz_gpu));
}

void fdtdCudaUnifiedMemory(size_t NX, size_t NY, size_t tmax, DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz,
    ResultDatabase &DB, OptionParser &op)
{
    const bool uvm = op.getOptionBool("uvm");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));

    double t_start, t_end;

    DATA_TYPE *_fict_gpu;
    DATA_TYPE *ex_gpu;
    DATA_TYPE *ey_gpu;
    DATA_TYPE *hz_gpu;

    _fict_gpu = _fict_;
    ex_gpu = ex;
    ey_gpu = ey;
    hz_gpu = hz;

    if (uvm)
    {
        // Do nothing
    }
    else if (uvm_advise)
    {
        checkCudaErrors(hipMemAdvise(_fict_gpu, sizeof(DATA_TYPE) * tmax, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(_fict_gpu, sizeof(DATA_TYPE) * tmax, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(_fict_gpu, sizeof(DATA_TYPE) * tmax, hipMemAdviseSetAccessedBy, device));

        checkCudaErrors(hipMemAdvise(ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemAdviseSetAccessedBy, device));

        checkCudaErrors(hipMemAdvise(ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemAdviseSetAccessedBy, device));

        checkCudaErrors(hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemAdviseSetAccessedBy, device));
    }
    else if (uvm_prefetch)
    {
        checkCudaErrors(hipMemPrefetchAsync(_fict_gpu, sizeof(DATA_TYPE) * tmax, device));
        checkCudaErrors(hipMemPrefetchAsync(ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1), device));
        checkCudaErrors(hipMemPrefetchAsync(ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY, device));
        checkCudaErrors(hipMemPrefetchAsync(hz_gpu,  sizeof(DATA_TYPE) * NX * NY, device));
    }
    else if (uvm_prefetch_advise)
    {
        checkCudaErrors(hipMemAdvise(_fict_gpu, sizeof(DATA_TYPE) * tmax, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(_fict_gpu, sizeof(DATA_TYPE) * tmax, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(_fict_gpu, sizeof(DATA_TYPE) * tmax, hipMemAdviseSetAccessedBy, device));

        checkCudaErrors(hipMemAdvise(ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemAdviseSetAccessedBy, device));

        checkCudaErrors(hipMemAdvise(ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemAdviseSetAccessedBy, device));

        checkCudaErrors(hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemAdviseSetAccessedBy, device));

        checkCudaErrors(hipMemPrefetchAsync(_fict_gpu, sizeof(DATA_TYPE) * tmax, device));
        checkCudaErrors(hipMemPrefetchAsync(ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1), device));
        checkCudaErrors(hipMemPrefetchAsync(ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY, device));
        checkCudaErrors(hipMemPrefetchAsync(hz_gpu,  sizeof(DATA_TYPE) * NX * NY, device));
    }
    else
    {
        std::cerr << "unrecognized uvm flag, exiting..." << std::endl;
        exit(-1);
    }

    dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
    dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)), (size_t)ceil(((float)NX) / ((float)block.y)));

    // hipStream_t stream1, stream2;
    // checkCudaErrors(hipStreamCreate(&stream1));
    // checkCudaErrors(hipStreamCreate(&stream2));
    t_start = rtclock();

    if (op.getOptionBool("coop"))
    {
        fdtd_params params;
        params.NX = NX;
        params.NY = NY;
        params._fict_ = _fict_gpu;
        params.ex = ex_gpu;
        params.ey = ey_gpu;
        params.hz = hz_gpu;
        void *p_params = {&params};
        for (int t = 0; t < tmax; t++)
        {
            params.t = t;
            checkCudaErrors(hipLaunchCooperativeKernel((void*)fdtd_coop_kernel, grid, block, &p_params));
        }
    }
    else
    {
        hipStream_t stream1, stream2;
        checkCudaErrors(hipStreamCreate(&stream1));
        checkCudaErrors(hipStreamCreate(&stream2));
        for (int t = 0; t < tmax; t++)
        {
            fdtd_step1_kernel<<<grid,block,0,stream1>>>(NX, NY, _fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
            fdtd_step2_kernel<<<grid,block,0,stream1>>>(NX, NY, ex_gpu, ey_gpu, hz_gpu, t);
            fdtd_step3_kernel<<<grid,block>>>(NX, NY, ex_gpu, ey_gpu, hz_gpu, t);
        }
        checkCudaErrors(hipStreamDestroy(stream1));
        checkCudaErrors(hipStreamDestroy(stream2));
    }

    hipDeviceSynchronize();
    t_end = rtclock();
    fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise)
    {
    // Do nothing
    }

    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise)
    {
        checkCudaErrors(hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemAdviseSetReadMostly, hipCpuDeviceId));
        checkCudaErrors(hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
        checkCudaErrors(hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemAdviseSetAccessedBy, hipCpuDeviceId));
        checkCudaErrors(hipMemPrefetchAsync(hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipCpuDeviceId));
        checkCudaErrors(hipDeviceSynchronize());
    }
}

void addBenchmarkSpecOptions(OptionParser &op)
{
    op.addOption("uvm", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory, only demand paging");
    op.addOption("uvm-advise", OPT_BOOL, "0", "guide the driver about memory usage patterns");
    op.addOption("uvm-prefetch", OPT_BOOL, "0", "prefetch memory the specified destination device");
    op.addOption("uvm-prefetch-advise", OPT_BOOL, "0", "prefetch memory the specified destination device with memory guidance on");
    op.addOption("coop", OPT_BOOL, "0", "use cooperative kernel instead normal kernels");
    op.addOption("compare", OPT_BOOL, "0", "compare GPU output with CPU output");
}

void RunBenchmark(ResultDatabase &DB, OptionParser &op)
{
    const bool uvm = op.getOptionBool("uvm");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
    const bool compare = op.getOptionBool("compare");

    const size_t s = 5;
    size_t NX_sizes[s] = {100, 1000, 2000, 8000, 16000};
    size_t NY_sizes[s] = {200, 1200, 2600, 9600, 20000};
    size_t tmax_sizes[s] =  {240, 500, 1000, 4000, 8000};

    size_t NX = NX_sizes[op.getOptionInt("size") - 1];
    size_t NY = NY_sizes[op.getOptionInt("size") - 1];
    size_t tmax = tmax_sizes[op.getOptionInt("size") - 1];

    double t_start, t_end;

    DATA_TYPE* _fict_;
    DATA_TYPE* ex;
    DATA_TYPE* ey;
    DATA_TYPE* hz;
    DATA_TYPE* hz_outputFromGpu;

    if (compare)
    {
        if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise)
        {
            DATA_TYPE* _fict_gpu;
            DATA_TYPE* ex_gpu;
            DATA_TYPE* ey_gpu;
            DATA_TYPE* hz_gpu;
            checkCudaErrors(hipMallocManaged(&_fict_gpu, tmax*sizeof(DATA_TYPE)));
            checkCudaErrors(hipMallocManaged(&ex_gpu, NX*(NY+1)*sizeof(DATA_TYPE)));
            checkCudaErrors(hipMallocManaged(&ey_gpu, (NX+1)*NY*sizeof(DATA_TYPE)));
            checkCudaErrors(hipMallocManaged(&hz_gpu, NX*NY*sizeof(DATA_TYPE)));

            _fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
            assert(_fict_);
            ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
            assert(ex);
            ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
            assert(ey);
            hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
            assert(hz);

            init_arrays(NX, NY, tmax, _fict_gpu, ex_gpu, ey_gpu, hz_gpu);
            checkCudaErrors(hipMemcpy(_fict_, _fict_gpu, tmax*sizeof(DATA_TYPE), hipMemcpyHostToHost));
            checkCudaErrors(hipMemcpy(ex, ex_gpu, NX*(NY+1)*sizeof(DATA_TYPE), hipMemcpyHostToHost));
            checkCudaErrors(hipMemcpy(ey, ey_gpu, (NX+1)*NY*sizeof(DATA_TYPE), hipMemcpyHostToHost));
            checkCudaErrors(hipMemcpy(hz, hz_gpu, NX*NY*sizeof(DATA_TYPE), hipMemcpyHostToHost));
            
            fdtdCudaUnifiedMemory(NX, NY, tmax, _fict_gpu, ex_gpu, ey_gpu, hz_gpu, DB, op);
            t_start = rtclock();
            runFdtd(NX, NY, tmax, _fict_, ex, ey, hz);
            t_end = rtclock();
            fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
            compareResults(NX, NY, hz, hz_gpu);

            checkCudaErrors(hipFree(_fict_gpu));
            checkCudaErrors(hipFree(ex_gpu));
            checkCudaErrors(hipFree(ey_gpu));
            checkCudaErrors(hipFree(hz_gpu));
            free(_fict_);
            free(ex);
            free(ey);
            free(hz);
        }
        else
        {
            _fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
            assert(_fict_);
            ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
            assert(ex);
            ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
            assert(ey);
            hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
            assert(hz);
            hz_outputFromGpu = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
            assert(hz_outputFromGpu);

            init_arrays(NX, NY, tmax, _fict_, ex, ey, hz);
            fdtdCuda(NX, NY, tmax, _fict_, ex, ey, hz, hz_outputFromGpu, DB, op);
            t_start = rtclock();
            runFdtd(NX, NY, tmax, _fict_, ex, ey, hz);
            t_end = rtclock();
            fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
            compareResults(NX, NY, hz, hz_outputFromGpu);

            free(_fict_);
            free(ex);
            free(ey);
            free(hz);
            free(hz_outputFromGpu);
        }
    }
    else
    {
        if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise)
        {
            checkCudaErrors(hipMallocManaged(&_fict_, tmax*sizeof(DATA_TYPE)));
            checkCudaErrors(hipMallocManaged(&ex, NX*(NY+1)*sizeof(DATA_TYPE)));
            checkCudaErrors(hipMallocManaged(&ey, (NX+1)*NY*sizeof(DATA_TYPE)));
            checkCudaErrors(hipMallocManaged(&hz, NX*NY*sizeof(DATA_TYPE)));

            init_arrays(NX, NY, tmax, _fict_, ex, ey, hz);
            fdtdCudaUnifiedMemory(NX, NY, tmax, _fict_, ex, ey, hz, DB, op);

            checkCudaErrors(hipFree(_fict_));
            checkCudaErrors(hipFree(ex));
            checkCudaErrors(hipFree(ey));
            checkCudaErrors(hipFree(hz));
        }
        else
        {
            _fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
            assert(_fict_);
            ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
            assert(ex);
            ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
            assert(ey);
            hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
            assert(hz);
            hz_outputFromGpu = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
            assert(hz_outputFromGpu);

            init_arrays(NX, NY, tmax, _fict_, ex, ey, hz);
            fdtdCuda(NX, NY, tmax, _fict_, ex, ey, hz, hz_outputFromGpu, DB, op);

            free(_fict_);
            free(ex);
            free(ey);
            free(hz);
            free(hz_outputFromGpu);
        }
    }
}
