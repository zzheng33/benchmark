#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\level2\where\where.cu
//
// summary:	Where class
// 
// origin: 
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"
#include <stdio.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
    
/// <summary>	The kernel time. </summary>
float kernelTime = 0.0f;
/// <summary>	The transfer time. </summary>
float transferTime = 0.0f;

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Gets the stop. </summary>
///
/// <value>	The stop. </value>
////////////////////////////////////////////////////////////////////////////////////////////////////

hipEvent_t start, stop;
/// <summary>	The elapsed time. </summary>
float elapsedTime;

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Checks. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/20/2020. </remarks>
///
/// <param name="val">  	The value. </param>
/// <param name="bound">	The bound. </param>
///
/// <returns>	True if it succeeds, false if it fails. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ bool check(int val, int bound) {
    return (val < bound);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Mark matches. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/20/2020. </remarks>
///
/// <param name="arr">	  	[in,out] If non-null, the array. </param>
/// <param name="results">	[in,out] If non-null, the results. </param>
/// <param name="size">   	The size. </param>
/// <param name="bound">  	The bound. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void markMatches(int *arr, int *results, int size, int bound) {

    // Block index
    int bx = blockIdx.x;

    // Thread index
    int tx = threadIdx.x;

    int tid = (blockDim.x * bx) + tx;

    for( ; tid < size; tid += blockDim.x * gridDim.x) {
        if(check(arr[tid], bound)) {
            results[tid] = 1;
        } else {
            results[tid] = 0;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Map matches. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/20/2020. </remarks>
///
/// <param name="arr">	  	[in,out] If non-null, the array. </param>
/// <param name="results">	[in,out] If non-null, the results. </param>
/// <param name="prefix"> 	[in,out] If non-null, the prefix. </param>
/// <param name="final">  	[in,out] If non-null, the final. </param>
/// <param name="size">   	The size. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void mapMatches(int *arr, int *results, int *prefix, int *final, int size) {

    // Block index
    int bx = blockIdx.x;

    // Thread index
    int tx = threadIdx.x;

    int tid = (blockDim.x * bx) + tx;

    for( ; tid < size; tid += blockDim.x * gridDim.x) {
        if(results[tid]) {
            final[prefix[tid]] = arr[tid];
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Seed array. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/20/2020. </remarks>
///
/// <param name="arr"> 	[in,out] If non-null, the array. </param>
/// <param name="size">	The size. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void seedArr(int *arr, int size) {
    for(int i = 0; i < size; i++) {
        arr[i] = rand() % 100;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Wheres. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/20/2020. </remarks>
///
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="size">	   	The size. </param>
/// <param name="coverage">	The coverage. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void where(ResultDatabase &resultDB, OptionParser &op, int size, int coverage) {
    const bool uvm = op.getOptionBool("uvm");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));

    int *arr = NULL;
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        checkCudaErrors(hipMallocManaged(&arr, sizeof(int) * size));
    } else {
        arr = (int*)malloc(sizeof(int) * size);
        assert(arr);
    }
    int *final;
    seedArr(arr, size);

    int *d_arr;
    int *d_results;
    int *d_prefix;
    int *d_final;
    
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        d_arr = arr;
        checkCudaErrors(hipMallocManaged( (void**) &d_results, sizeof(int) * size));
        checkCudaErrors(hipMallocManaged( (void**) &d_prefix, sizeof(int) * size));
    } else {
        checkCudaErrors(hipMalloc( (void**) &d_arr, sizeof(int) * size));
        checkCudaErrors(hipMalloc( (void**) &d_results, sizeof(int) * size));
        checkCudaErrors(hipMalloc( (void**) &d_prefix, sizeof(int) * size));
    }

    checkCudaErrors(hipEventRecord(start, 0));
    if (uvm) {
        // do nothing
    } else if (uvm_advise) {
        checkCudaErrors(hipMemAdvise(d_arr, sizeof(int) * size, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(d_arr, sizeof(int) * size, hipMemAdviseSetPreferredLocation, device));
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(d_arr, sizeof(int) * size, device));
    } else if (uvm_prefetch_advise) {
        checkCudaErrors(hipMemAdvise(d_arr, sizeof(int) * size, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(d_arr, sizeof(int) * size, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemPrefetchAsync(d_arr, sizeof(int) * size, device));
    } else {
        checkCudaErrors(hipMemcpy(d_arr, arr, sizeof(int) * size, hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3;

    dim3 grid(size / 1024 + 1, 1, 1);
    dim3 threads(1024, 1, 1);
    checkCudaErrors(hipEventRecord(start, 0));
    markMatches<<<grid, threads>>>(d_arr, d_results, size, coverage);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    kernelTime += elapsedTime * 1.e-3;
    CHECK_CUDA_ERROR();

    checkCudaErrors(hipEventRecord(start, 0));
    thrust::exclusive_scan(thrust::device, d_results, d_results + size, d_prefix);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    kernelTime += elapsedTime * 1.e-3;
    CHECK_CUDA_ERROR();

    int matchSize;
    checkCudaErrors(hipEventRecord(start, 0));
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        matchSize = (int)*(d_prefix + size - 1);
    } else {
        checkCudaErrors(hipMemcpy(&matchSize, d_prefix + size - 1, sizeof(int), hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3;
    matchSize++;

    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        checkCudaErrors(hipMallocManaged( (void**) &d_final, sizeof(int) * matchSize));
        final = d_final;
    } else {
        checkCudaErrors(hipMalloc( (void**) &d_final, sizeof(int) * matchSize));
        final = (int*)malloc(sizeof(int) * matchSize);
        assert(final);
    }

    checkCudaErrors(hipEventRecord(start, 0));
    mapMatches<<<grid, threads>>>(d_arr, d_results, d_prefix, d_final, size);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    kernelTime += elapsedTime * 1.e-3;
    CHECK_CUDA_ERROR();

    checkCudaErrors(hipEventRecord(start, 0));
    // No cpy just demand paging
    if (uvm) {
        // Do nothing
    } else if (uvm_advise) {
        checkCudaErrors(hipMemAdvise(final, sizeof(int) * matchSize, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(final, sizeof(int) * matchSize, hipMemAdviseSetPreferredLocation, device));
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(final, sizeof(int) * matchSize, hipCpuDeviceId));
    } else if (uvm_prefetch_advise) {
        checkCudaErrors(hipMemAdvise(final, sizeof(int) * matchSize, hipMemAdviseSetReadMostly, device));
        checkCudaErrors(hipMemAdvise(final, sizeof(int) * matchSize, hipMemAdviseSetPreferredLocation, device));
        checkCudaErrors(hipMemPrefetchAsync(final, sizeof(int) * matchSize, hipCpuDeviceId));
    } else {
        checkCudaErrors(hipMemcpy(final, d_final, sizeof(int) * matchSize, hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3;

    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        checkCudaErrors(hipFree(d_arr));
        checkCudaErrors(hipFree(d_results));
        checkCudaErrors(hipFree(d_prefix));
        checkCudaErrors(hipFree(d_final));
    } else {
        free(arr);
        free(final);
        checkCudaErrors(hipFree(d_arr));
        checkCudaErrors(hipFree(d_results));
        checkCudaErrors(hipFree(d_prefix));
        checkCudaErrors(hipFree(d_final));
    }
    
    char atts[1024];
    sprintf(atts, "size:%d, coverage:%d", size, coverage);
    resultDB.AddResult("where_kernel_time", atts, "sec", kernelTime);
    resultDB.AddResult("where_transfer_time", atts, "sec", transferTime);
    resultDB.AddResult("where_total_time", atts, "sec", kernelTime+transferTime);
    resultDB.AddResult("where_parity", atts, "N", transferTime / kernelTime);
    resultDB.AddOverall("Time", "sec", kernelTime+transferTime);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Adds a benchmark specifier options. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/20/2020. </remarks>
///
/// <param name="op">	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void addBenchmarkSpecOptions(OptionParser &op) {
  op.addOption("length", OPT_INT, "0", "number of elements in input");
  op.addOption("coverage", OPT_INT, "-1", "0 to 100 percentage of elements to allow through where filter");
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the benchmark operation. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu), 5/20/2020. </remarks>
///
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="op">	   	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
    printf("Running Where\n");

    srand(7);

    bool quiet = op.getOptionBool("quiet");
    int size = op.getOptionInt("length");
    int coverage = op.getOptionInt("coverage");
    if (size == 0 || coverage == -1) {
        int sizes[5] = {1000, 10000, 500000000, 1000000000, 1050000000};
        int coverages[5] = {20, 30, 40, 80, 240};
        size = sizes[op.getOptionInt("size") - 1];
        coverage = coverages[op.getOptionInt("size") - 1];
    }

    if (!quiet) {
        printf("Using size=%d, coverage=%d\n", size, coverage);
    }

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    int passes = op.getOptionInt("passes");
    for (int i = 0; i < passes; i++) {
        kernelTime = 0.0f;
        transferTime = 0.0f;
        if(!quiet) {
            printf("Pass %d: ", i);
        }
        where(resultDB, op, size, coverage);
        if(!quiet) {
            printf("Done.\n");
        }
    }
}
