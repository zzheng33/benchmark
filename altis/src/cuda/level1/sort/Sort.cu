#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\level1\sort\Sort.cu
//
// summary:	Sort class
// 
// origin: SHOC Benchmark (https://github.com/vetter/shoc)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"
#include "Sort.h"
#include "sort_kernel.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/types.h>
#include <vector>

#define SEED 7

using namespace std;

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op) {}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the radix sort benchmark
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parsefilePathr / parameter database
//
// Returns:  nothing, results are stored in resultDB
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications: Bodun Hu
// Add UVM support
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
    cout << "Running Sort" << endl;
  srand(SEED);
  bool quiet = op.getOptionBool("quiet");
  const bool uvm = op.getOptionBool("uvm");
  const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
  const bool uvm_advise = op.getOptionBool("uvm-advise");
  const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
  int device = 0;
  checkCudaErrors(hipGetDevice(&device));

  // Determine size of the array to sort
  int size;
  long long bytes;
  string filePath = op.getOptionString("inputFile");
  ifstream inputFile(filePath.c_str());
  if (filePath == "") {
    if(!quiet) {
        printf("Using problem size %d\n", (int)op.getOptionInt("size"));
    }
    int probSizes[5] = {32, 64, 256, 512, 1024};
    size = probSizes[op.getOptionInt("size") - 1] * 1024 * 1024;
  } else {
    inputFile >> size;
  }
  bytes = size * sizeof(uint);
  if(!quiet) {
    printf("Size: %d items, Bytes: %lld\n", size, bytes);
  }

  // If input file given, populate array
  uint *sourceInput = (uint *)malloc(bytes);
  if (filePath != "") {
      for (int i = 0; i < size; i++) {
          inputFile >> sourceInput[i];
      }
  }

  // create input data on CPU
  uint *hKeys = NULL;
  uint *hVals = NULL;

  ////////////////////////////////////////////////////////////////////////////////////////////////////
  /// <summary>	allocate using UVM API. </summary>
  ///
  /// <remarks>	Ed, 5/20/2020. </remarks>
  ///
  ////////////////////////////////////////////////////////////////////////////////////////////////////

  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
      checkCudaErrors(hipMallocManaged(&hKeys, bytes));
      checkCudaErrors(hipMallocManaged(&hVals, bytes));
  } else {
      checkCudaErrors(hipHostMalloc((void **)&hKeys, bytes));
      checkCudaErrors(hipHostMalloc((void **)&hVals, bytes));
  }

  // Allocate space for block sums in the scan kernel.
  uint numLevelsAllocated = 0;
  uint maxNumScanElements = size;
  uint numScanElts = maxNumScanElements;
  uint level = 0;

  do {
    uint numBlocks =
        max(1, (int)ceil((float)numScanElts / (4 * SCAN_BLOCK_SIZE)));
    if (numBlocks > 1) {
      level++;
    }
    numScanElts = numBlocks;
  } while (numScanElts > 1);

  uint **scanBlockSums = NULL;
  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
      checkCudaErrors(hipMallocManaged(&scanBlockSums, (level+1) * sizeof(uint *)));
  } else {
      scanBlockSums = (uint **)malloc((level + 1) * sizeof(uint *));
      assert(scanBlockSums);
  }

  numLevelsAllocated = level + 1;
  numScanElts = maxNumScanElements;
  level = 0;

  do {
    uint numBlocks =
        max(1, (int)ceil((float)numScanElts / (4 * SCAN_BLOCK_SIZE)));
    if (numBlocks > 1) {
      // Malloc device mem for block sums
      if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
          checkCudaErrors(hipMallocManaged((void **)&(scanBlockSums[level]),
                                      numBlocks * sizeof(uint)));
      } else {
          checkCudaErrors(hipMalloc((void **)&(scanBlockSums[level]),
                                      numBlocks * sizeof(uint)));
      }
      level++;
    }
    numScanElts = numBlocks;
  } while (numScanElts > 1);

  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    checkCudaErrors(hipMallocManaged((void **)&(scanBlockSums[level]), sizeof(uint)));
  } else {
    checkCudaErrors(hipMalloc((void **)&(scanBlockSums[level]), sizeof(uint)));
  }

  // Allocate device mem for sorting kernels
  uint *dKeys, *dVals, *dTempKeys, *dTempVals;

  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    dKeys = hKeys;
    dVals = hVals;
    checkCudaErrors(hipMallocManaged((void **)&dTempKeys, bytes));
    checkCudaErrors(hipMallocManaged((void **)&dTempVals, bytes));
  } else {
    checkCudaErrors(hipMalloc((void **)&dKeys, bytes));
    checkCudaErrors(hipMalloc((void **)&dVals, bytes));
    checkCudaErrors(hipMalloc((void **)&dTempKeys, bytes));
    checkCudaErrors(hipMalloc((void **)&dTempVals, bytes));
  }

  // Each thread in the sort kernel handles 4 elements
  size_t numSortGroups = size / (4 * SORT_BLOCK_SIZE);

  uint *dCounters, *dCounterSums, *dBlockOffsets;
  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    checkCudaErrors(hipMallocManaged((void **)&dCounters,
                              WARP_SIZE * numSortGroups * sizeof(uint)));
    checkCudaErrors(hipMallocManaged((void **)&dCounterSums,
                              WARP_SIZE * numSortGroups * sizeof(uint)));
    checkCudaErrors(hipMallocManaged((void **)&dBlockOffsets,
                              WARP_SIZE * numSortGroups * sizeof(uint)));
  }
  else {
    checkCudaErrors(hipMalloc((void **)&dCounters,
                              WARP_SIZE * numSortGroups * sizeof(uint)));
    checkCudaErrors(hipMalloc((void **)&dCounterSums,
                              WARP_SIZE * numSortGroups * sizeof(uint)));
    checkCudaErrors(hipMalloc((void **)&dBlockOffsets,
                              WARP_SIZE * numSortGroups * sizeof(uint)));
  }

  int iterations = op.getOptionInt("passes");
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  for (int it = 0; it < iterations; it++) {
    if(!quiet) {
        printf("Pass %d: ", it);
    }
/// <summary>	Initialize host memory to some pattern. </summary>
    for (uint i = 0; i < size; i++) {
      hKeys[i] = i % 1024;
      if (filePath == "") {
        hVals[i] = rand() % 1024;
      } else {
        hVals[i] = sourceInput[i];
      }
    }

    // Copy inputs to GPU
    double transferTime = 0.;
    checkCudaErrors(hipEventRecord(start, 0));
    if (uvm) {
      // do nothing
    } else if (uvm_advise) {
      checkCudaErrors(hipMemAdvise(dKeys, bytes, hipMemAdviseSetPreferredLocation, device));
      checkCudaErrors(hipMemAdvise(dVals, bytes, hipMemAdviseSetPreferredLocation, device));
    } else if (uvm_prefetch) {
      checkCudaErrors(hipMemPrefetchAsync(dKeys, bytes, device));
      hipStream_t s1;
      checkCudaErrors(hipStreamCreate(&s1));
      checkCudaErrors(hipMemPrefetchAsync(dVals, bytes, device, s1));
      checkCudaErrors(hipStreamDestroy(s1));
    } else if (uvm_prefetch_advise) {
      checkCudaErrors(hipMemAdvise(dKeys, bytes, hipMemAdviseSetPreferredLocation, device));
      checkCudaErrors(hipMemAdvise(dVals, bytes, hipMemAdviseSetPreferredLocation, device));
      checkCudaErrors(hipMemPrefetchAsync(dKeys, bytes, device));
      hipStream_t s1;
      checkCudaErrors(hipStreamCreate(&s1));
      checkCudaErrors(hipMemPrefetchAsync(dVals, bytes, device, (hipStream_t)1));
      checkCudaErrors(hipStreamDestroy(s1));
    } else {
        checkCudaErrors(hipMemcpy(dKeys, hKeys, bytes, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dVals, hVals, bytes, hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3; // convert to seconds

    checkCudaErrors(hipEventRecord(start, 0));
    // Perform Radix Sort (4 bits at a time)
    for (int i = 0; i < SORT_BITS; i += 4) {
      radixSortStep(4, i, (uint4 *)dKeys, (uint4 *)dVals, (uint4 *)dTempKeys,
                    (uint4 *)dTempVals, dCounters, dCounterSums, dBlockOffsets,
                    scanBlockSums, size);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    double kernelTime = elapsedTime * 1.e-3;
    // Readback data from device
    checkCudaErrors(hipEventRecord(start, 0));

    // prefetch or demand paging
    if (uvm) {
      // do nothing
    } else if (uvm_advise) {
      checkCudaErrors(hipMemAdvise(dKeys, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
      checkCudaErrors(hipMemAdvise(dVals, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
    } else if (uvm_prefetch) {
      checkCudaErrors(hipMemPrefetchAsync(dKeys, bytes, hipCpuDeviceId));
      checkCudaErrors(hipMemPrefetchAsync(dVals, bytes, hipCpuDeviceId, (hipStream_t)1));
    } else if (uvm_prefetch_advise) {
      checkCudaErrors(hipMemAdvise(dKeys, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
      checkCudaErrors(hipMemAdvise(dVals, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
      checkCudaErrors(hipMemPrefetchAsync(dKeys, bytes, hipCpuDeviceId));
      checkCudaErrors(hipMemPrefetchAsync(dVals, bytes, hipCpuDeviceId, (hipStream_t)1));
    } else {
      checkCudaErrors(hipMemcpy(hKeys, dKeys, bytes, hipMemcpyDeviceToHost));
      checkCudaErrors(hipMemcpy(hVals, dVals, bytes, hipMemcpyDeviceToHost));
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3;

    // Test to make sure data was sorted properly, if not, return
    if (!verifySort(hKeys, hVals, size, op.getOptionBool("verbose"), op.getOptionBool("quiet"))) {
      return;
    }

    char atts[1024];
    sprintf(atts, "%ditems", size);
    double gb = (bytes * 2.) / (1000. * 1000. * 1000.);
    resultDB.AddResult("Sort-KernelTime", atts, "sec", kernelTime);
    resultDB.AddResult("Sort-TransferTime", atts, "sec", transferTime);
    resultDB.AddResult("Sort-TotalTime", atts, "sec", transferTime + kernelTime);
    resultDB.AddResult("Sort-Rate", atts, "GB/s", gb / kernelTime);
    resultDB.AddResult("Sort-Rate_PCIe", atts, "GB/s",
                       gb / (kernelTime + transferTime));
    resultDB.AddResult("Sort-Rate_Parity", atts, "N",
                       transferTime / kernelTime);
    resultDB.AddOverall("Rate", "GB/s", gb/kernelTime);
  }
  // Clean up
  for (int i = 0; i < numLevelsAllocated; i++) {
    checkCudaErrors(hipFree(scanBlockSums[i]));
  }
  checkCudaErrors(hipFree(dKeys));
  checkCudaErrors(hipFree(dVals));
  checkCudaErrors(hipFree(dTempKeys));
  checkCudaErrors(hipFree(dTempVals));
  checkCudaErrors(hipFree(dCounters));
  checkCudaErrors(hipFree(dCounterSums));
  checkCudaErrors(hipFree(dBlockOffsets));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    checkCudaErrors(hipFree(scanBlockSums));
  } else {
    free(scanBlockSums);
    checkCudaErrors(hipHostFree(hKeys));
    checkCudaErrors(hipHostFree(hVals));
  }
  free(sourceInput);
}

// ****************************************************************************
// Function: radixSortStep
//
// Purpose:
//   This function performs a radix sort, using bits startbit to
//   (startbit + nbits).  It is designed to sort by 4 bits at a time.
//   It also reorders the data in the values array based on the sort.
//
// Arguments:
//      nbits: the number of key bits to use
//      startbit: the bit to start on, 0 = lsb
//      keys: the input array of keys
//      values: the input array of values
//      tempKeys: temporary storage, same size as keys
//      tempValues: temporary storage, same size as values
//      counters: storage for the index counters, used in sort
//      countersSum: storage for the sum of the counters
//      blockOffsets: storage used in sort
//      scanBlockSums: input to Scan, see below
//      numElements: the number of elements to sort
//
// Returns: nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// origin: SHOC (https://github.com/vetter/shoc)
//
// ****************************************************************************
void radixSortStep(uint nbits, uint startbit, uint4 *keys, uint4 *values,
                   uint4 *tempKeys, uint4 *tempValues, uint *counters,
                   uint *countersSum, uint *blockOffsets, uint **scanBlockSums,
                   uint numElements) {
  // Threads handle either 4 or two elements each
  const size_t radixGlobalWorkSize = numElements / 4;
  const size_t findGlobalWorkSize = numElements / 2;
  const size_t reorderGlobalWorkSize = numElements / 2;

  // Radix kernel uses block size of 128, others use 256 (same as scan)
  const size_t radixBlocks = radixGlobalWorkSize / SORT_BLOCK_SIZE;
  const size_t findBlocks = findGlobalWorkSize / SCAN_BLOCK_SIZE;
  const size_t reorderBlocks = reorderGlobalWorkSize / SCAN_BLOCK_SIZE;

  radixSortBlocks<<<radixBlocks, SORT_BLOCK_SIZE,
                    4 * sizeof(uint) * SORT_BLOCK_SIZE>>>(
      nbits, startbit, tempKeys, tempValues, keys, values);

  findRadixOffsets<<<findBlocks, SCAN_BLOCK_SIZE,
                     2 * SCAN_BLOCK_SIZE * sizeof(uint)>>>(
      (uint2 *)tempKeys, counters, blockOffsets, startbit, numElements,
      findBlocks);

  scanArrayRecursive(countersSum, counters, 16 * reorderBlocks, 0,
                     scanBlockSums);

  reorderData<<<reorderBlocks, SCAN_BLOCK_SIZE>>>(
      startbit, (uint *)keys, (uint *)values, (uint2 *)tempKeys,
      (uint2 *)tempValues, blockOffsets, countersSum, counters, reorderBlocks);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Perform scan op on input array recursively. </summary>
///
/// <remarks>	Ed, 5/19/2020. </remarks>
///
/// <param name="outArray">   	[in,out] If non-null, array of outs. </param>
/// <param name="inArray">	  	[in,out] If non-null, array of INS. </param>
/// <param name="numElements">	Number of elements. </param>
/// <param name="level">	  	The num of levels. </param>
/// <param name="blockSums">  	[in,out] The block sum array. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void scanArrayRecursive(uint *outArray, uint *inArray, int numElements,
                        int level, uint **blockSums) {
  // Kernels handle 8 elems per thread
  unsigned int numBlocks =
      max(1, (unsigned int)ceil((float)numElements / (4.f * SCAN_BLOCK_SIZE)));
  unsigned int sharedEltsPerBlock = SCAN_BLOCK_SIZE * 2;
  unsigned int sharedMemSize = sizeof(uint) * sharedEltsPerBlock;

  bool fullBlock = (numElements == numBlocks * 4 * SCAN_BLOCK_SIZE);

  dim3 grid(numBlocks, 1, 1);
  dim3 threads(SCAN_BLOCK_SIZE, 1, 1);

  // execute the scan
  if (numBlocks > 1) {
    scan<<<grid, threads, sharedMemSize>>>(outArray, inArray, blockSums[level],
                                           numElements, fullBlock, true);
  } else {
    scan<<<grid, threads, sharedMemSize>>>(outArray, inArray, blockSums[level],
                                           numElements, fullBlock, false);
  }
  if (numBlocks > 1) {
    scanArrayRecursive(blockSums[level], blockSums[level], numBlocks, level + 1,
                       blockSums);
    vectorAddUniform4<<<grid, threads>>>(outArray, blockSums[level],
                                         numElements);
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Verify the correctness of sort on cpu. </summary>
///
/// <remarks>	Kyle Spafford, 8/13/2009
/// 			Ed, 5/19/2020. </remarks>
///
/// <param name="keys">   	[in,out] If non-null, the keys. </param>
/// <param name="vals">   	[in,out] If non-null, the vals. </param>
/// <param name="size">   	The size. </param>
/// <param name="verbose">	True to verbose. </param>
/// <param name="quiet">  	True to quiet. </param>
///
/// <returns>	True if it succeeds, false if it fails. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

bool verifySort(uint *keys, uint *vals, const size_t size, bool verbose, bool quiet) {
  bool passed = true;
  for (unsigned int i = 0; i < size - 1; i++) {
    if (keys[i] > keys[i + 1]) {
      passed = false;
      if(verbose && !quiet)  {
          cout << "Failure: at idx: " << i << endl;
          cout << "Key: " << keys[i] << " Val: " << vals[i] << endl;
          cout << "Idx: " << i + 1 << " Key: " << keys[i + 1]
              << " Val: " << vals[i + 1] << endl;
      }
    }
  }
  if (!quiet) {
      cout << "Test ";
      if (passed) {
          cout << "Passed" << endl;
      } else {
          cout << "Failed" << endl;
      }
  }
  return passed;
}
