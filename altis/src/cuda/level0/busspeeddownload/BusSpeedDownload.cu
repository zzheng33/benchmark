////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\level0\busspeeddownload\BusSpeedDownload.cu
//
// summary:	Bus speed download test.
// 
// modified from: SHOC Benchmark Suite (https://github.com/vetter/shoc)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"
#include <stdio.h>

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific command line argument parsing.
//
//   -nopinned
//   This option controls whether page-locked or "pinned" memory is used.
//   The use of pinned memory typically results in higher bandwidth for data
//   transfer between host and device.
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation: September 08, 2009
// 
// Modifications: Ed, 5/19/2020.
//
// ****************************************************************************

void addBenchmarkSpecOptions(OptionParser &op) {
    op.addOption("uvm-prefetch", OPT_BOOL, "0", "prefetch memory the specified destination device");
    op.addOption("pinned", OPT_BOOL, "0", "use pinned (pagelocked) memory");
}

// ****************************************************************************
// Function: runBenchmark
//
// Purpose:
//   Measures the bandwidth of the bus connecting the host processor to the
//   OpenCL device.  This benchmark repeatedly transfers data chunks of various
//   sizes across the bus to the OpenCL device, and calculates the bandwidth.
//
//
// Arguments:
//  resultDB: the benchmark stores its results in this ResultDatabase
//  op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation: September 08, 2009
//
// Modifications:
//    Jeremy Meredith, Wed Dec  1 17:05:27 EST 2010
//    Added calculation of latency estimate.
//  
//    Bodun Hu (bodunhu@utexas.edu), Jan 3 2021
//    Added UVM prefetch.
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
    cout << "Running BusSpeedDownload" << endl;
    const bool verbose = op.getOptionBool("verbose");
    const bool quiet = op.getOptionBool("quiet");
    const bool pinned = op.getOptionBool("pinned");

    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");

    // Sizes are in kb
    int nSizes = 21;
    int sizes[21] = {1,     2,     4,     8,      16,     32,    64,
                    128,   256,   512,   1024,   2048,   4096,  8192,
                    16384, 32768, 65536, 131072, 262144, 524288, 1048576};
    long long numMaxFloats = 1024 * (sizes[nSizes - 1]) / 4;

    // Create some host memory pattern
    float *hostMem = NULL;
    if (uvm_prefetch) {
        hipMallocManaged((void **)&hostMem, sizeof(float) * numMaxFloats);
        while (hipGetLastError() != hipSuccess) {
            // drop the size and try again
            if (verbose && !quiet) {
                cout << " - dropping size allocating unified mem\n";
            }
            --nSizes;
            if (nSizes < 1) {
                cerr << "Error: Couldn't allocated any unified buffer\n";
                return;
            }
            numMaxFloats = 1024 * (sizes[nSizes - 1]) / 4;
            hipMallocManaged((void **)&hostMem, sizeof(float) * numMaxFloats);
        }
    } else {
        if (pinned) {
            hipHostMalloc((void **)&hostMem, sizeof(float) * numMaxFloats);
            while (hipGetLastError() != hipSuccess) {
                // drop the size and try again
                if (verbose && !quiet) {
                    cout << " - dropping size allocating pinned mem\n";
                }
                --nSizes;
                if (nSizes < 1) {
                    cerr << "Error: Couldn't allocated any pinned buffer\n";
                    return;
                }
                numMaxFloats = 1024 * (sizes[nSizes - 1]) / 4;
                hipHostMalloc((void **)&hostMem, sizeof(float) * numMaxFloats);
            }
        } else {
            hostMem = new float[numMaxFloats];
        }
    }

    // Initialize host memory
    for (int i = 0; i < numMaxFloats; i++) {
        hostMem[i] = i % 77;
    }

    float *device = NULL;
    if (uvm_prefetch) {
        device = hostMem;
    } else {
        hipMalloc((void **)&device, sizeof(float) * numMaxFloats);
        while (hipGetLastError() != hipSuccess) {
            // drop the size and try again
            if (verbose && !quiet) {
                cout << " - dropping size allocating device mem\n";
            }
            --nSizes;
            if (nSizes < 1) {
                cerr << "Error: Couldn't allocated any device buffer\n";
                return;
            }
            numMaxFloats = 1024 * (sizes[nSizes - 1]) / 4;
            hipMalloc((void **)&device, sizeof(float) * numMaxFloats);
        }
    }

    const unsigned int passes = op.getOptionInt("passes");

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    int deviceID = 0;
    checkCudaErrors(hipGetDevice(&deviceID));

    // Three passes, forward and backward both
    for (int pass = 0; pass < passes; pass++) {
        // store the times temporarily to estimate latency
        // float times[nSizes];
        // Step through sizes forward on even passes and backward on odd
        for (int i = 0; i < nSizes; i++) {
            int sizeIndex;
            if ((pass % 2) == 0)
                sizeIndex = i;
            else
                sizeIndex = (nSizes - 1) - i;

            int nbytes = sizes[sizeIndex] * 1024;

            hipEventRecord(start, 0);
            if (uvm_prefetch) {
                // Use default stream
                checkCudaErrors(hipMemPrefetchAsync(device, nbytes, deviceID));
                checkCudaErrors(hipStreamSynchronize(0));
            } else {
                checkCudaErrors(hipMemcpy(device, hostMem, nbytes, hipMemcpyHostToDevice));
            }
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float t = 0;
            hipEventElapsedTime(&t, start, stop);
            // times[sizeIndex] = t;

            // Convert to GB/sec
            if (verbose && !quiet) {
                cout << "size " << sizes[sizeIndex] << "k took " << t << " ms\n";
            }

            double speed = (double(sizes[sizeIndex]) * 1024. / (1000 * 1000)) / t;
            resultDB.AddResult("DownloadSpeed", "---", "GB/sec", speed);
            resultDB.AddOverall("DownloadSpeed", "GB/sec", speed);

            // Move data back to host if it's already prefetched to device
            if (uvm_prefetch) {
                checkCudaErrors(hipMemPrefetchAsync(device, nbytes, hipCpuDeviceId));
                checkCudaErrors(hipStreamSynchronize(0));
            }
        }
    }

    // Cleanup
    if (uvm_prefetch) {
        checkCudaErrors(hipFree((void *)device));
    } else {
        checkCudaErrors(hipFree((void *)device));
        if (pinned) {
            checkCudaErrors(hipHostFree((void *)hostMem));
        } else {
            delete[] hostMem;
        }
    }
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
}
