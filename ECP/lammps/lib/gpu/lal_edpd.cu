#include "hip/hip_runtime.h"
// **************************************************************************
//                                   edpd.cu
//                             -------------------
//                           Trung Dac Nguyen (U Chicago)
//
//  Device code for acceleration of the edpd pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : September 2023
//    email                : ndactrung@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( vel_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture_2d( vel_tex,int4);
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#define EPSILON (numtyp)1.0e-10

//#define _USE_UNIFORM_SARU_LCG
//#define _USE_UNIFORM_SARU_TEA8
//#define _USE_GAUSSIAN_SARU_LCG

#if !defined(_USE_UNIFORM_SARU_LCG) && !defined(_USE_UNIFORM_SARU_TEA8) && !defined(_USE_GAUSSIAN_SARU_LCG)
#define _USE_UNIFORM_SARU_LCG
#endif

// References:
// 1. Y. Afshar, F. Schmid, A. Pishevar, S. Worley, Comput. Phys. Comm. 184 (2013), 1119–1128.
// 2. C. L. Phillips, J. A. Anderson, S. C. Glotzer, Comput. Phys. Comm. 230 (2011), 7191-7201.
// PRNG period = 3666320093*2^32 ~ 2^64 ~ 10^19

#define LCGA 0x4beb5d59 /* Full period 32 bit LCG */
#define LCGC 0x2600e1f7
#define oWeylPeriod 0xda879add /* Prime period 3666320093 */
#define oWeylOffset 0x8009d14b
#define TWO_N32 0.232830643653869628906250e-9f /* 2^-32 */

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0]
// using the inherent LCG, then multiply u with sqrt(3) to "match"
// with a normal random distribution.
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
// Curly brackets to make variables local to the scope.
#ifdef _USE_UNIFORM_SARU_LCG
#define SQRT3 (numtyp)1.7320508075688772935274463
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  state = LCGA*state + LCGC;                                                  \
  wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod);   \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0] using TEA8
// then multiply u with sqrt(3) to "match" with a normal random distribution
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
#ifdef _USE_UNIFORM_SARU_TEA8
#define SQRT3 (numtyp)1.7320508075688772935274463
#define k0 0xA341316C
#define k1 0xC8013EA4
#define k2 0xAD90777D
#define k3 0x7E95761E
#define delta 0x9e3779b9
#define rounds 8
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int sum = 0;                                                       \
  for (int i=0; i < rounds; i++) {                                            \
    sum += delta;                                                             \
    state += ((wstate<<4) + k0)^(wstate + sum)^((wstate>>5) + k1);            \
    wstate += ((state<<4) + k2)^(state + sum)^((state>>5) + k3);              \
  }                                                                           \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns two uniformly distributed random numbers r1 and r2 in [-1.0;1.0],
// and uses the polar method (Marsaglia's) to transform to a normal random value
// This is used to compared with CPU DPD using RandMars::gaussian()
#ifdef _USE_GAUSSIAN_SARU_LCG
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state=0x12345678;                                              \
  unsigned int wstate=12345678;                                               \
  state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));                      \
  wstate = (state + seed2) ^ (((signed int)state)>>8);                        \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int v, s;                                                          \
  numtyp r1, r2, rsq;                                                         \
  while (1) {                                                                 \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r1 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r2 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    rsq = r1 * r1 + r2 * r2;                                                  \
    if (rsq < (numtyp)1.0) break;                                             \
  }                                                                           \
  numtyp fac = ucl_sqrt((numtyp)-2.0*log(rsq)/rsq);                           \
  randnum = r2*fac;                                                           \
}
#endif

#if (SHUFFLE_AVAIL == 0)

#define store_heatflux(Qi, ii, inum, tid, t_per_atom, offset, Q)             \
  if (t_per_atom>1) {                                                        \
    simdsync();                                                              \
    simd_reduce_add1(t_per_atom, red_acc, offset, tid, Qi);                  \
  }                                                                          \
  if (offset==0 && ii<inum) {                                                \
    Q[ii]=Qi;                                                                \
  }
#else
#define store_heatflux(Qi, ii, inum, tid, t_per_atom, offset, Q)             \
  if (t_per_atom>1) {                                                        \
    simd_reduce_add1(t_per_atom,Qi);                                         \
  }                                                                          \
  if (offset==0 && ii<inum) {                                                \
    Q[ii]=Qi;                                                                \
  }
#endif

#define MIN(A,B) ((A) < (B) ? (A) : (B))
#define MAX(A,B) ((A) < (B) ? (B) : (A))

// note the change in coeff: coeff.x = a0, coeff.y = gamma, coeff.z = cut (no sigma)

__kernel void k_edpd(const __global numtyp4 *restrict x_,
                     const __global numtyp4 *restrict extra,
                     const __global numtyp4 *restrict coeff,
                     const __global numtyp4 *restrict coeff2,
                     const __global numtyp *restrict mass,
                     const __global numtyp4 *restrict sc,
                     const __global numtyp4 *restrict kc,
                     const int lj_types,
                     const __global numtyp *restrict sp_lj,
                     const __global numtyp *restrict sp_sqrt,
                     const __global int * dev_nbor,
                     const __global int * dev_packed,
                     __global acctyp3 *restrict ans,
                     __global acctyp *restrict engv,
                     __global acctyp *restrict Q,
                     const int eflag, const int vflag,
                     const int power_flag, const int kappa_flag,
                     const int inum, const int nbor_pitch,
                     const __global numtyp4 *restrict v_,
                     const __global numtyp *restrict cutsq,
                     const numtyp dtinvsqrt, const int seed,
                     const int timestep, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }
  acctyp Qi = (acctyp)0;

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp mass_itype = mass[itype];
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    const numtyp4 Tcvi = extra[i];
    numtyp Ti = Tcvi.x;
    numtyp cvi = Tcvi.y;

    numtyp factor_dpd;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      factor_dpd = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;
        numtyp vijeij = dot*rinv;

        const numtyp coeffx=coeff[mtype].x; // a0[itype][jtype]
        const numtyp coeffy=coeff[mtype].y; // gamma[itype][jtype]
        const numtyp coeffz=coeff[mtype].z; // cut[itype][jtype]

        const numtyp4 Tcvj = extra[j];
        numtyp Tj = Tcvj.x;
        numtyp cvj = Tcvj.y;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }

        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        numtyp T_ij=(numtyp)0.5*(Ti+Tj);
        numtyp4 T_pow;
        T_pow.x = T_ij - (numtyp)1.0;
        T_pow.y = T_pow.x*T_pow.x;
        T_pow.z = T_pow.x*T_pow.y;
        T_pow.w = T_pow.x*T_pow.z;

        numtyp coeff2x = coeff2[mtype].x; //power[itype][jtype]
        numtyp coeff2y = coeff2[mtype].y; //kappa[itype][jtype]
        numtyp coeff2z = coeff2[mtype].z; //powerT[itype][jtype]
        numtyp coeff2w = coeff2[mtype].w; //cutT[itype][jtype]
        numtyp power_d = coeff2x;
        if (power_flag) {
          numtyp factor = (numtyp)1.0;
          factor += sc[mtype].x*T_pow.x + sc[mtype].y*T_pow.y +
            sc[mtype].z*T_pow.z + sc[mtype].w*T_pow.w;
          power_d *= factor;
        }

        power_d = MAX((numtyp)0.01,power_d);
        numtyp wc = (numtyp)1.0 - r/coeffz; // cut[itype][jtype]
        wc = MAX((numtyp)0.0,MIN((numtyp)1.0,wc));
        numtyp wr = ucl_pow(wc, (numtyp)0.5*power_d);

        numtyp kboltz = (numtyp)1.0;
        numtyp GammaIJ = coeffy; // gamma[itype][jtype]
        numtyp SigmaIJ = (numtyp)4.0*GammaIJ*kboltz*Ti*Tj/(Ti+Tj);
        SigmaIJ = ucl_sqrt(SigmaIJ);

        numtyp force =  coeffx*T_ij*wc; // a0[itype][jtype]
        force -= GammaIJ *wr*wr *dot*rinv;
        force += SigmaIJ * wr *randnum * dtinvsqrt;
        force *= factor_dpd*rinv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // heat transfer

        if (r < coeff2w) {
          numtyp wrT = (numtyp)1.0 - r/coeff2w;
          wrT = MAX((numtyp)0.0,MIN((numtyp)1.0,wrT));
          wrT = ucl_pow(wrT, (numtyp)0.5*coeff2z); // powerT[itype][jtype]
          numtyp randnumT = (numtyp)0;
          saru(tag1, tag2, seed+tag1+tag2, timestep, randnumT); // randomT->gaussian();
          randnumT = MAX((numtyp)-5.0,MIN(randnum,(numtyp)5.0));

          numtyp kappaT = coeff2y; // kappa[itype][jtype]
          if (kappa_flag) {
            numtyp factor = (numtyp)1.0;
            factor += kc[mtype].x*T_pow.x + kc[mtype].y*T_pow.y +
              kc[mtype].z*T_pow.z + kc[mtype].w*T_pow.w;
            kappaT *= factor;
          }

          numtyp kij = cvi*cvj*kappaT * T_ij*T_ij;
          numtyp alphaij = ucl_sqrt((numtyp)2.0*kboltz*kij);

          numtyp dQc = kij * wrT*wrT * (Tj - Ti)/(Ti*Tj);
          numtyp dQd = wr*wr*( GammaIJ * vijeij*vijeij - SigmaIJ*SigmaIJ/mass_itype ) - SigmaIJ * wr *vijeij *randnum;
          dQd /= (cvi+cvj);
          numtyp dQr = alphaij * wrT * dtinvsqrt * randnumT;
          Qi += (dQc + dQd + dQr );
        }

        if (EVFLAG && eflag) {
          numtyp e = (numtyp)0.5*coeffx*T_ij*coeffz * wc*wc;
          energy+=factor_dpd*e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
  store_heatflux(Qi,ii,inum,tid,t_per_atom,offset,Q);
}

__kernel void k_edpd_fast(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict extra,
                          const __global numtyp4 *restrict coeff_in,
                          const __global numtyp4 *restrict coeff2_in,
                          const __global numtyp *restrict mass,
                          const __global numtyp4 *restrict sc_in,
                          const __global numtyp4 *restrict kc_in,
                          const __global numtyp *restrict sp_lj_in,
                          const __global numtyp *restrict sp_sqrt_in,
                          const __global int * dev_nbor,
                          const __global int * dev_packed,
                          __global acctyp3 *restrict ans,
                          __global acctyp *restrict engv,
                          __global acctyp *restrict Q,
                          const int eflag, const int vflag,
                          const int power_flag, const int kappa_flag,
                          const int inum, const int nbor_pitch,
                          const __global numtyp4 *restrict v_,
                          const __global numtyp *restrict cutsq,
                          const numtyp dtinvsqrt, const int seed,
                          const int timestep, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 sc[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 kc[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4) {
    sp_lj[tid]=sp_lj_in[tid];
  }
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
    coeff2[tid]=coeff2_in[tid];
    sc[tid]=sc_in[tid];
    kc[tid]=kc_in[tid];
  }
  __syncthreads();
  #else
  const numtyp coeffx=coeff_in[ONETYPE].x;   // a0[itype][jtype]
  const numtyp coeffy=coeff_in[ONETYPE].y;   // gamma[itype][jtype]
  const numtyp coeffz=coeff_in[ONETYPE].z;   // cut[itype][jtype]
  const numtyp coeff2x=coeff2_in[ONETYPE].x; // power[itype][jtype]
  const numtyp coeff2y=coeff2_in[ONETYPE].y; // kappa[itype][jtype]
  const numtyp coeff2z=coeff2_in[ONETYPE].z; // powerT[itype][jtype]
  const numtyp coeff2w=coeff2_in[ONETYPE].w; // cutT[itype][jtype]
  const numtyp cutsq_p=cutsq[ONETYPE];
  const numtyp scx=sc_in[ONETYPE].x;
  const numtyp scy=sc_in[ONETYPE].y;
  const numtyp scz=sc_in[ONETYPE].z;
  const numtyp scw=sc_in[ONETYPE].w;
  const numtyp kcx=kc_in[ONETYPE].x;
  const numtyp kcy=kc_in[ONETYPE].y;
  const numtyp kcz=kc_in[ONETYPE].z;
  const numtyp kcw=kc_in[ONETYPE].w;
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }
  acctyp Qi = (acctyp)0;

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    numtyp mass_itype = mass[iw];
    #ifndef ONETYPE
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    #endif
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    const numtyp4 Tcvi = extra[i];
    numtyp Ti = Tcvi.x;
    numtyp cvi = Tcvi.y;

    #ifndef ONETYPE
    numtyp factor_dpd;
    #endif
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      #ifndef ONETYPE
      factor_dpd = sp_lj[sbmask(j)];
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      const numtyp cutsq_p=cutsq[mtype];
      #endif
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq_p) {
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;
        numtyp vijeij = dot*rinv;

        #ifndef ONETYPE
        const numtyp coeffx=coeff[mtype].x;   // a0[itype][jtype]
        const numtyp coeffy=coeff[mtype].y;   // gamma[itype][jtype]
        const numtyp coeffz=coeff[mtype].z;   // cut[itype][jtype]
        const numtyp coeff2x=coeff2[mtype].x; // power[itype][jtype]
        const numtyp coeff2y=coeff2[mtype].y; // kappa[itype][jtype]
        const numtyp coeff2z=coeff2[mtype].z; // powerT[itype][jtype]
        const numtyp coeff2w=coeff2[mtype].w; // cutT[itype][jtype]
        const numtyp scx = sc[mtype].x;
        const numtyp scy = sc[mtype].y;
        const numtyp scz = sc[mtype].z;
        const numtyp scw = sc[mtype].w;
        const numtyp kcx = kc[mtype].x;
        const numtyp kcy = kc[mtype].y;
        const numtyp kcz = kc[mtype].z;
        const numtyp kcw = kc[mtype].w;
        #endif

        const numtyp4 Tcvj = extra[j];
        numtyp Tj = Tcvj.x;
        numtyp cvj = Tcvj.y;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }
        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        numtyp T_ij=(numtyp)0.5*(Ti+Tj);
        numtyp4 T_pow;
        T_pow.x = T_ij - (numtyp)1.0;
        T_pow.y = T_pow.x*T_pow.x;
        T_pow.z = T_pow.x*T_pow.y;
        T_pow.w = T_pow.x*T_pow.z;

        numtyp power_d = coeff2x; // power[itype][jtype]
        if (power_flag) {
          numtyp factor = (numtyp)1.0;
          factor += scx*T_pow.x + scy*T_pow.y + scz*T_pow.z + scw*T_pow.w;
          power_d *= factor;
        }

        power_d = MAX((numtyp)0.01,power_d);
        numtyp wc = (numtyp)1.0 - r/coeffz; // cut[itype][jtype]
        wc = MAX((numtyp)0.0,MIN((numtyp)1.0,wc));
        numtyp wr = ucl_pow((numtyp)wc, (numtyp)0.5*power_d);

        numtyp kboltz = (numtyp)1.0;
        numtyp GammaIJ = coeffy; // gamma[itype][jtype]
        numtyp SigmaIJ = (numtyp)4.0*GammaIJ*kboltz*Ti*Tj/(Ti+Tj);
        SigmaIJ = ucl_sqrt(SigmaIJ);

        numtyp force =  coeffx*T_ij*wc; // a0[itype][jtype]
        force -= GammaIJ *wr*wr *dot*rinv;
        force += SigmaIJ* wr *randnum * dtinvsqrt;
        #ifndef ONETYPE
        force *= factor_dpd*rinv;
        #else
        force *= rinv;
        #endif

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // heat transfer

        if (r < coeff2w) {
          numtyp wrT = (numtyp)1.0 - r/coeff2w;
          wrT = MAX((numtyp)0.0,MIN((numtyp)1.0,wrT));
          wrT = ucl_pow(wrT, (numtyp)0.5*coeff2z); // powerT[itype][jtype]
          numtyp randnumT = (numtyp)0;
          saru(tag1, tag2, seed+tag1+tag2, timestep, randnumT); // randomT->gaussian();
          randnumT = MAX((numtyp)-5.0,MIN(randnum,(numtyp)5.0));

          numtyp kappaT = coeff2y; // kappa[itype][jtype]
          if (kappa_flag) {
            numtyp factor = (numtyp)1.0;
            factor += kcx*T_pow.x +  kcy*T_pow.y + kcz*T_pow.z + kcw*T_pow.w;
            kappaT *= factor;
          }

          numtyp kij = cvi*cvj*kappaT * T_ij*T_ij;
          numtyp alphaij = ucl_sqrt((numtyp)2.0*kboltz*kij);

          numtyp dQc = kij * wrT*wrT * (Tj - Ti )/(Ti*Tj);
          numtyp dQd = wr*wr*( GammaIJ * vijeij*vijeij - SigmaIJ*SigmaIJ/mass_itype ) - SigmaIJ * wr *vijeij *randnum;
          dQd /= (cvi+cvj);
          numtyp dQr = alphaij * wrT * dtinvsqrt * randnumT;
          Qi += (dQc + dQd + dQr );
        }

        if (EVFLAG && eflag) {
          numtyp e = (numtyp)0.5*coeffx*T_ij*coeffz * wc*wc;
          #ifndef ONETYPE
          energy+=factor_dpd*e;
          #else
          energy+=e;
          #endif
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }

      }
    } // for nbor
  } // if ii

  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag, ans,engv);
  store_heatflux(Qi,ii,inum,tid,t_per_atom,offset,Q);
}

