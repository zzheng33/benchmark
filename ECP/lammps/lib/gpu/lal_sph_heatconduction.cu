#include "hip/hip_runtime.h"
// **************************************************************************
//                             sph_heatconduction.cu
//                             ---------------------
//                           Trung Dac Nguyen (U Chicago)
//
//  Device code for acceleration of the sph/heatconduction pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : September 2023
//    email                : ndactrung@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( vel_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture_2d( vel_tex,int4);
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#if (SHUFFLE_AVAIL == 0)

#define store_dE(dEacc, ii, inum, tid, t_per_atom, offset, i, dE)           \
  if (t_per_atom>1) {                                                       \
    simdsync();                                                             \
    simd_reduce_add1(t_per_atom, red_acc, offset, tid, dEacc);              \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    dE[i]=dEacc;                                                            \
  }
#else
#define store_drhoE(dEacc, ii, inum, tid, t_per_atom, offset, i, dE)        \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      dEacc += shfl_down(dEacc, s, t_per_atom);                             \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    dE[i]=dEacc;                                                           \
  }
#endif

/* ------------------------------------------------------------------------ */

__kernel void k_sph_heatconduction(const __global numtyp4 *restrict x_,
                       const __global numtyp4 *restrict extra,
                       const __global numtyp4 *restrict coeff,
                       const __global numtyp *restrict mass,
                       const int lj_types,
                       const __global numtyp *restrict sp_lj,
                       const __global int * dev_nbor,
                       const __global int * dev_packed,
                       __global acctyp3 *restrict ans,
                       __global acctyp *restrict engv,
                       __global acctyp *restrict dE,
                       const int eflag, const int vflag,
                       const int inum, const int nbor_pitch,
                       const __global numtyp4 *restrict v_,
                       const int dimension, const int t_per_atom) {
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
#if (SHUFFLE_AVAIL == 0)
  local_allocate_store_pair();
#endif

  acctyp dEacc = (acctyp)0;

  if (ii<inum) {
    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp mass_itype = mass[itype];

    const numtyp4 extrai = extra[i];
    numtyp rhoi = extrai.x;
    numtyp esphi = extrai.y;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<coeff[mtype].z) { // cutsq[itype][jtype]
        numtyp mass_jtype = mass[jtype];
        const numtyp coeffx=coeff[mtype].x;  // alpha[itype][jtype]
        const numtyp coeffy=coeff[mtype].y;  // cut[itype][jtype]

        const numtyp4 extraj = extra[j];
        numtyp rhoj = extraj.x;
        numtyp esphj = extraj.y;

        numtyp h = coeffy; // cut[itype][jtype]
        numtyp ih = ucl_recip(h); // (numtyp)1.0 / h;
        numtyp ihsq = ih * ih;

        numtyp wfd = h - ucl_sqrt(rsq);
        if (dimension == 3) {
          // Lucy Kernel, 3d
          wfd = (numtyp)-25.066903536973515383 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
          // Lucy Kernel, 2d
          wfd = (numtyp)-19.098593171027440292 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        // total thermal energy increment
        numtyp D = coeffx; // alpha[itype][jtype]  diffusion coefficient
        numtyp deltaE = (numtyp)2.0 * mass_itype * mass_jtype / (mass_itype + mass_jtype);
        deltaE *= (rhoi + rhoj) / (rhoi * rhoj);
        deltaE *= D * (esphi - esphj) * wfd;

        // change in thermal energy, desph[i]
        dEacc += deltaE;

      }
    } // for nbor
  } // if ii

  store_drhoE(dEacc,ii,inum,tid,t_per_atom,offset,i,dE);
}

__kernel void k_sph_heatconduction_fast(const __global numtyp4 *restrict x_,
                            const __global numtyp4 *restrict extra,
                            const __global numtyp4 *restrict coeff_in,
                            const __global numtyp *restrict mass,
                            const __global numtyp *restrict sp_lj_in,
                            const __global int * dev_nbor,
                            const __global int * dev_packed,
                            __global acctyp3 *restrict ans,
                            __global acctyp *restrict engv,
                            __global acctyp *restrict dE,
                            const int eflag, const int vflag,
                            const int inum, const int nbor_pitch,
                            const __global numtyp4 *restrict v_,
                            const int dimension, const int t_per_atom) {
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
  }
  __syncthreads();
  #else
  const numtyp coeffx=coeff_in[ONETYPE].x;   // alpha[itype][jtype]
  const numtyp coeffy=coeff_in[ONETYPE].y;   // cut[itype][jtype]
  const numtyp cutsq_p=coeff_in[ONETYPE].z;  // cutsq[itype][jtype]
  #endif

  int n_stride;
#if (SHUFFLE_AVAIL == 0)
  local_allocate_store_pair();
#endif

  acctyp dEacc = (acctyp)0;

  if (ii<inum) {
    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    numtyp mass_itype = mass[iw];
    #ifndef ONETYPE
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    #endif

    const numtyp4 extrai = extra[i];
    numtyp rhoi = extrai.x;
    numtyp esphi = extrai.y;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      #ifndef ONETYPE
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      const numtyp cutsq_p=coeff[mtype].z;
      #endif

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq_p) {
        numtyp mass_jtype = mass[jtype];
        #ifndef ONETYPE
        const numtyp coeffx=coeff[mtype].x;  // alpha[itype][jtype]
        const numtyp coeffy=coeff[mtype].y;  // cut[itype][jtype]
        #endif
        const numtyp4 extraj = extra[j];
        numtyp rhoj = extraj.x;
        numtyp esphj = extraj.y;

        numtyp h = coeffy; // cut[itype][jtype]
        numtyp ih = ih = ucl_recip(h); // (numtyp)1.0 / h;
        numtyp ihsq = ih * ih;

        numtyp wfd = h - ucl_sqrt(rsq);
        if (dimension == 3) {
          // Lucy Kernel, 3d
          wfd = (numtyp)-25.066903536973515383 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
          // Lucy Kernel, 2d
          wfd = (numtyp)-19.098593171027440292 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        // total thermal energy increment
        numtyp D = coeffx; // alpha[itype][jtype]  diffusion coefficient
        numtyp deltaE = (numtyp)2.0 * mass_itype * mass_jtype / (mass_itype + mass_jtype);
        deltaE *= (rhoi + rhoj) / (rhoi * rhoj);
        deltaE *= D * (esphi - esphj) * wfd;

        // change in thermal energy, desph[i]
        dEacc += deltaE;

      }
    } // for nbor
  } // if ii

  store_drhoE(dEacc,ii,inum,tid,t_per_atom,offset,i,dE);
}

