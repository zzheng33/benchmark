#include "hip/hip_runtime.h"
// **************************************************************************
//                              sph_taitwater.cu
//                             -------------------
//                           Trung Dac Nguyen (U Chicago)
//
//  Device code for acceleration of the sph/taitwater pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : September 2023
//    email                : ndactrung@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( vel_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture_2d( vel_tex,int4);
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#if (SHUFFLE_AVAIL == 0)

#define store_drhoE(drhoEacc, ii, inum, tid, t_per_atom, offset, i, drhoE)  \
  if (t_per_atom>1) {                                                       \
    simdsync();                                                             \
    simd_reduce_add2(t_per_atom, red_acc, offset, tid,                      \
                     drhoEacc.x, drhoEacc.y);                               \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    drhoE[i]=drhoEacc.x;                                                    \
    drhoE[i+inum]=drhoEacc.y;                                               \
  }
#else
#define store_drhoE(drhoEacc, ii, inum, tid, t_per_atom, offset, i, drhoE)  \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      drhoEacc.x += shfl_down(drhoEacc.x, s, t_per_atom);                   \
      drhoEacc.y += shfl_down(drhoEacc.y, s, t_per_atom);                   \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    drhoE[i]=drhoEacc.x;                                                    \
    drhoE[i+inum]=drhoEacc.y;                                               \
  }
#endif

__kernel void k_sph_taitwater(const __global numtyp4 *restrict x_,
                              const __global numtyp4 *restrict extra,
                              const __global numtyp4 *restrict coeff,
                              const __global numtyp4 *restrict coeff2,
                              const int lj_types,
                              const __global numtyp *restrict sp_lj,
                              const __global int * dev_nbor,
                              const __global int * dev_packed,
                              __global acctyp3 *restrict ans,
                              __global acctyp *restrict engv,
                              __global acctyp *restrict drhoE,
                              const int eflag, const int vflag,
                              const int inum, const int nbor_pitch,
                              const __global numtyp4 *restrict v_,
                              const int dimension, const int t_per_atom) {
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }
  acctyp2 drhoEacc;
  drhoEacc.x = drhoEacc.y = (acctyp)0;

  if (ii<inum) {
    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp mass_itype = coeff2[itype].x;
    numtyp rho0_itype = coeff2[itype].y;
    numtyp soundspeed_itype = coeff2[itype].z;
    numtyp B_itype = coeff2[itype].w;
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];

    const numtyp4 extrai = extra[i];
    numtyp rhoi = extrai.x;

    // compute pressure of atom i with Tait EOS
    numtyp tmp = rhoi / rho0_itype;
    numtyp fi = tmp * tmp * tmp;
    fi = B_itype * (fi * fi * tmp - (numtyp)1.0);
    fi /= (rhoi * rhoi);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<coeff[mtype].z) { // cutsq[itype][jtype]
        const numtyp coeffx=coeff[mtype].x;  // viscosity[itype][jtype]
        const numtyp coeffy=coeff[mtype].y;  // cut[itype][jtype]

        numtyp mass_jtype = coeff2[jtype].x;
        numtyp rho0_jtype = coeff2[jtype].y;
        numtyp soundspeed_jtype = coeff2[jtype].z;
        numtyp B_jtype = coeff2[jtype].w;

        const numtyp4 extraj = extra[j];
        numtyp rhoj = extraj.x;

        numtyp h = coeffy; // cut[itype][jtype]
        numtyp ih = ucl_recip(h); // (numtyp)1.0 / h;
        numtyp ihsq = ih * ih;

        numtyp wfd = h - ucl_sqrt(rsq);
        if (dimension == 3) {
          // Lucy Kernel, 3d
          wfd = (numtyp)-25.066903536973515383 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
          // Lucy Kernel, 2d
          wfd = (numtyp)-19.098593171027440292 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        // compute pressure  of atom j with Tait EOS

        numtyp tmp = rhoj / rho0_jtype;
        numtyp fj = tmp * tmp * tmp;
        fj = B_jtype * (fj * fj * tmp - (numtyp)1.0);
        fj /= (rhoj * rhoj);

        // dot product of velocity delta and distance vector
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp delVdotDelR = delx*delvx + dely*delvy + delz*delvz;

        // artificial viscosity (Monaghan 1992)
        numtyp fvisc = (numtyp)0;
        if (delVdotDelR < (numtyp)0) {
          numtyp mu = h * delVdotDelR / (rsq + (numtyp)0.01 * h * h);
          fvisc = -coeffx * (soundspeed_itype + soundspeed_jtype) * mu / (rhoi + rhoj);
        } 

        // total pair force & thermal energy increment
        numtyp force = -mass_itype * mass_jtype * (fi + fj + fvisc) * wfd;
        numtyp deltaE = (numtyp)-0.5 * force * delVdotDelR;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // and change in density, drho[i]
        drhoEacc.x += mass_jtype * delVdotDelR * wfd;

        // change in thermal energy, desph[i]
        drhoEacc.y += deltaE;

        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
  store_drhoE(drhoEacc,ii,inum,tid,t_per_atom,offset,i,drhoE);
}

__kernel void k_sph_taitwater_fast(const __global numtyp4 *restrict x_,
                                   const __global numtyp4 *restrict extra,
                                   const __global numtyp4 *restrict coeff_in,
                                   const __global numtyp4 *restrict coeff2_in,
                                   const __global numtyp *restrict sp_lj_in,
                                   const __global int * dev_nbor,
                                   const __global int * dev_packed,
                                   __global acctyp3 *restrict ans,
                                   __global acctyp *restrict engv,
                                   __global acctyp *restrict drhoE,
                                   const int eflag, const int vflag,
                                   const int inum, const int nbor_pitch,
                                   const __global numtyp4 *restrict v_,
                                   const int dimension, const int t_per_atom) {
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES) {
    coeff2[tid] = coeff2_in[tid];
  }
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
  }
  __syncthreads();
  #else
  const numtyp coeffx=coeff_in[ONETYPE].x;   // viscosity[itype][jtype]
  const numtyp coeffy=coeff_in[ONETYPE].y;   // cut[itype][jtype]
  const numtyp cutsq_p=coeff_in[ONETYPE].z;  // cutsq[itype][jtype]
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }
  acctyp2 drhoEacc;
  drhoEacc.x = drhoEacc.y = (acctyp)0;

  if (ii<inum) {
    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    numtyp mass_itype = coeff2[iw].x;
    numtyp rho0_itype = coeff2[iw].y;
    numtyp soundspeed_itype = coeff2[iw].z;
    numtyp B_itype = coeff2[iw].w;
    #ifndef ONETYPE
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    #endif
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];

    const numtyp4 extrai = extra[i];
    numtyp rhoi = extrai.x;

    // compute pressure of atom i with Tait EOS
    numtyp tmp = rhoi / rho0_itype;
    numtyp fi = tmp * tmp * tmp;
    fi = B_itype * (fi * fi * tmp - (numtyp)1.0);
    fi /= (rhoi * rhoi);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      #ifndef ONETYPE
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      const numtyp cutsq_p=coeff[mtype].z;
      #endif
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq_p) {
        #ifndef ONETYPE
        const numtyp coeffx=coeff[mtype].x;  // viscosity[itype][jtype]
        const numtyp coeffy=coeff[mtype].y;  // cut[itype][jtype]
        #endif

        numtyp mass_jtype = coeff2[jtype].x;
        numtyp rho0_jtype = coeff2[jtype].y;
        numtyp soundspeed_jtype = coeff2[jtype].z;
        numtyp B_jtype = coeff2[jtype].w;

        const numtyp4 extraj = extra[j];
        numtyp rhoj = extraj.x;

        numtyp h = coeffy; // cut[itype][jtype]
        numtyp ih = ucl_recip(h); // (numtyp)1.0 / h;
        numtyp ihsq = ih * ih;

        numtyp wfd = h - ucl_sqrt(rsq);
        if (dimension == 3) {
          // Lucy Kernel, 3d
          wfd = (numtyp)-25.066903536973515383 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
          // Lucy Kernel, 2d
          wfd = (numtyp)-19.098593171027440292 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        // compute pressure  of atom j with Tait EOS
        numtyp tmp = rhoj / rho0_jtype;
        numtyp fj = tmp * tmp * tmp;
        fj = B_jtype * (fj * fj * tmp - (numtyp)1.0);
        fj /= (rhoj * rhoj);

        // dot product of velocity delta and distance vector
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp delVdotDelR = delx*delvx + dely*delvy + delz*delvz;

        // artificial viscosity (Monaghan 1992)
        numtyp fvisc = (numtyp)0;
        if (delVdotDelR < (numtyp)0) {
          numtyp mu = h * delVdotDelR / (rsq + (numtyp)0.01 * h * h);
          fvisc = -coeffx * (soundspeed_itype + soundspeed_jtype) * mu / (rhoi + rhoj);
        }

        // total pair force & thermal energy increment
        numtyp force = -mass_itype * mass_jtype * (fi + fj + fvisc) * wfd;
        numtyp deltaE = (numtyp)-0.5 * force * delVdotDelR;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // and change in density, drho[i]
        drhoEacc.x += mass_jtype * delVdotDelR * wfd;

        // change in thermal energy, desph[i]
        drhoEacc.y += deltaE;

        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii

  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag, ans,engv);
  store_drhoE(drhoEacc,ii,inum,tid,t_per_atom,offset,i,drhoE);
}

