#include "hip/hip_runtime.h"
// **************************************************************************
//                                   mdpd.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the mdpd pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : December 2023
//    email                : ndactrung@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( vel_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture_2d( vel_tex,int4);
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#define EPSILON (numtyp)1.0e-10

//#define _USE_UNIFORM_SARU_LCG
//#define _USE_UNIFORM_SARU_TEA8
//#define _USE_GAUSSIAN_SARU_LCG

#if !defined(_USE_UNIFORM_SARU_LCG) && !defined(_USE_UNIFORM_SARU_TEA8) && !defined(_USE_GAUSSIAN_SARU_LCG)
#define _USE_UNIFORM_SARU_LCG
#endif

// References:
// 1. Y. Afshar, F. Schmid, A. Pishevar, S. Worley, Comput. Phys. Comm. 184 (2013), 1119–1128.
// 2. C. L. Phillips, J. A. Anderson, S. C. Glotzer, Comput. Phys. Comm. 230 (2011), 7191-7201.
// PRNG period = 3666320093*2^32 ~ 2^64 ~ 10^19

#define LCGA 0x4beb5d59 /* Full period 32 bit LCG */
#define LCGC 0x2600e1f7
#define oWeylPeriod 0xda879add /* Prime period 3666320093 */
#define oWeylOffset 0x8009d14b
#define TWO_N32 0.232830643653869628906250e-9f /* 2^-32 */

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0]
// using the inherent LCG, then multiply u with sqrt(3) to "match"
// with a normal random distribution.
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
// Curly brackets to make variables local to the scope.
#ifdef _USE_UNIFORM_SARU_LCG
#define SQRT3 (numtyp)1.7320508075688772935274463
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  state = LCGA*state + LCGC;                                                  \
  wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod);   \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0] using TEA8
// then multiply u with sqrt(3) to "match" with a normal random distribution
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
#ifdef _USE_UNIFORM_SARU_TEA8
#define SQRT3 (numtyp)1.7320508075688772935274463
#define k0 0xA341316C
#define k1 0xC8013EA4
#define k2 0xAD90777D
#define k3 0x7E95761E
#define delta 0x9e3779b9
#define rounds 8
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int sum = 0;                                                       \
  for (int i=0; i < rounds; i++) {                                            \
    sum += delta;                                                             \
    state += ((wstate<<4) + k0)^(wstate + sum)^((wstate>>5) + k1);            \
    wstate += ((state<<4) + k2)^(state + sum)^((state>>5) + k3);              \
  }                                                                           \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns two uniformly distributed random numbers r1 and r2 in [-1.0;1.0],
// and uses the polar method (Marsaglia's) to transform to a normal random value
// This is used to compared with CPU DPD using RandMars::gaussian()
#ifdef _USE_GAUSSIAN_SARU_LCG
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state=0x12345678;                                              \
  unsigned int wstate=12345678;                                               \
  state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));                      \
  wstate = (state + seed2) ^ (((signed int)state)>>8);                        \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int v, s;                                                          \
  numtyp r1, r2, rsq;                                                         \
  while (1) {                                                                 \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r1 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r2 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    rsq = r1 * r1 + r2 * r2;                                                  \
    if (rsq < (numtyp)1.0) break;                                             \
  }                                                                           \
  numtyp fac = ucl_sqrt((numtyp)-2.0*log(rsq)/rsq);                           \
  randnum = r2*fac;                                                           \
}
#endif

#define MIN(A,B) ((A) < (B) ? (A) : (B))
#define MAX(A,B) ((A) < (B) ? (B) : (A))

// coeff.x = A_att, coeff.y = B_rep, coeff.z = gamma, coeff.w = sigma
// coeff2.x = cut, coeff2.y = cut_r, coeff2.z = cutsq

__kernel void k_mdpd(const __global numtyp4 *restrict x_,
                    const __global numtyp4 *restrict extra,
                    const __global numtyp4 *restrict coeff,
                    const __global numtyp4 *restrict coeff2,
                    const int lj_types,
                    const __global numtyp *restrict sp_lj,
                    const __global numtyp *restrict sp_sqrt,
                    const __global int * dev_nbor,
                    const __global int * dev_packed,
                    __global acctyp3 *restrict ans,
                    __global acctyp *restrict engv,
                    const int eflag, const int vflag, const int inum,
                    const int nbor_pitch,
                    const __global numtyp4 *restrict v_,
                    const __global numtyp *restrict cutsq,
                    const numtyp dtinvsqrt, const int seed,
                    const int timestep, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    const numtyp rhoi = extra[i].x;

    numtyp factor_dpd;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      factor_dpd = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<coeff2[mtype].z) {  // cutsq[itype][jtype]
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;

        numtyp A_attij = coeff[mtype].x;
        numtyp B_repij = coeff[mtype].y;
        numtyp gammaij = coeff[mtype].z;
        numtyp sigmaij = coeff[mtype].w;
        numtyp cutij =   coeff2[mtype].x;
        numtyp cut_rij = coeff2[mtype].y;

        numtyp wc = (numtyp)1.0 - r/cutij;
        numtyp wc_r = (numtyp)1.0 - r/cut_rij;
        wc_r = MAX(wc_r,(numtyp)0.0);
        numtyp wr = wc;

        const numtyp rhoj = extra[j].x;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }

        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        // conservative force = A_att * wc + B_rep*(rhoi+rhoj)*wc_r
        // drag force = -gamma * wr^2 * (delx dot delv) / r
        // random force = sigma * wr * rnd * dtinvsqrt;

        numtyp force = A_attij*wc + B_repij*(rhoi+rhoj)*wc_r;
        force -= gammaij*wr*wr*dot*rinv;
        force += sigmaij*wr*randnum*dtinvsqrt;
        force *= factor_dpd*rinv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          // unshifted eng of conservative term:
          // eng shifted to 0.0 at cutoff
          numtyp e = (numtyp)0.5*A_attij*cutij * wr*wr + (numtyp)0.5*B_repij*cut_rij*(rhoi+rhoj)*wc_r*wc_r;
          energy+=factor_dpd*e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

__kernel void k_mdpd_fast(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict extra,
                          const __global numtyp4 *restrict coeff_in,
                          const __global numtyp4 *restrict coeff2_in,
                          const __global numtyp *restrict sp_lj_in,
                          const __global numtyp *restrict sp_sqrt_in,
                          const __global int * dev_nbor,
                          const __global int * dev_packed,
                          __global acctyp3 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp4 *restrict v_,
                          const __global numtyp *restrict cutsq,
                          const numtyp dtinvsqrt, const int seed,
                          const int timestep, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4) {
    sp_lj[tid]=sp_lj_in[tid];
  }
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
    coeff2[tid]=coeff2_in[tid];
  }
  __syncthreads();
  #else
  const numtyp A_attij=coeff_in[ONETYPE].x;
  const numtyp B_repij=coeff_in[ONETYPE].y;
  const numtyp gammaij=coeff_in[ONETYPE].z;
  const numtyp sigmaij=coeff_in[ONETYPE].w;
  const numtyp cutij=coeff2_in[ONETYPE].x;
  const numtyp cut_rij=coeff2_in[ONETYPE].y;
  const numtyp cutsq_p=cutsq[ONETYPE];
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    #ifndef ONETYPE
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    #endif
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    const numtyp rhoi = extra[i].x;

    #ifndef ONETYPE
    numtyp factor_dpd;
    #endif
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      #ifndef ONETYPE
      factor_dpd = sp_lj[sbmask(j)];
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      const numtyp cutsq_p=cutsq[mtype];
      #endif
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq_p) {
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;

        #ifndef ONETYPE
        numtyp A_attij = coeff[mtype].x;
        numtyp B_repij = coeff[mtype].y;
        numtyp gammaij = coeff[mtype].z;
        numtyp sigmaij = coeff[mtype].w;
        numtyp cutij =   coeff2[mtype].x;
        numtyp cut_rij = coeff2[mtype].y;
        #endif

        numtyp wc = (numtyp)1.0 - r/cutij;
        numtyp wc_r = (numtyp)1.0 - r/cut_rij;
        wc_r = MAX(wc_r,(numtyp)0.0);
        numtyp wr = wc;

        const numtyp rhoj = extra[j].x;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }

        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        // conservative force = A_att * wc + B_rep*(rhoi+rhoj)*wc_r
        // drag force = -gamma * wr^2 * (delx dot delv) / r
        // random force = sigma * wr * rnd * dtinvsqrt;

        numtyp force = A_attij*wc + B_repij*(rhoi+rhoj)*wc_r;
        force -= gammaij*wr*wr*dot*rinv;
        force += sigmaij*wr*randnum*dtinvsqrt;
        #ifndef ONETYPE
        force *= factor_dpd*rinv;
        #else
        force*=rinv;
        #endif

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          // unshifted eng of conservative term:
          // eng shifted to 0.0 at cutoff
          numtyp e = (numtyp)0.5*A_attij*cutij * wr*wr + (numtyp)0.5*B_repij*cut_rij*(rhoi+rhoj)*wc_r*wc_r;
          #ifndef ONETYPE
          energy+=factor_dpd*e;
          #else
          energy+=e;
          #endif
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

