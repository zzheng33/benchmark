#include "hip/hip_runtime.h"
// **************************************************************************
//                                 sph_lj.cu
//                             -------------------
//                           Trung Dac Nguyen (U Chicago)
//
//  Device code for acceleration of the sph/lj pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : September 2023
//    email                : ndactrung@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( vel_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture_2d( vel_tex,int4);
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#if (SHUFFLE_AVAIL == 0)

#define store_drhoE(drhoEacc, ii, inum, tid, t_per_atom, offset, i, drhoE)  \
  if (t_per_atom>1) {                                                       \
    simdsync();                                                             \
    simd_reduce_add2(t_per_atom, red_acc, offset, tid,                      \
                     drhoEacc.x, drhoEacc.y);                               \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    drhoE[i]=drhoEacc.x;                                                    \
    drhoE[i+inum]=drhoEacc.y;                                               \
  }
#else
#define store_drhoE(drhoEacc, ii, inum, tid, t_per_atom, offset, i, drhoE)  \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      drhoEacc.x += shfl_down(drhoEacc.x, s, t_per_atom);                   \
      drhoEacc.y += shfl_down(drhoEacc.y, s, t_per_atom);                   \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    drhoE[i]=drhoEacc.x;                                                    \
    drhoE[i+inum]=drhoEacc.y;                                               \
  }
#endif

/* ------------------------------------------------------------------------ */
/* Lennard-Jones EOS,
   Francis H. Ree
   "Analytic representation of thermodynamic data for the Lennard‐Jones fluid",
   Journal of Chemical Physics 73 pp. 5401-5403 (1980)
   return p = pc[0], c = pc[1]
*/

ucl_inline void LJEOS2(const numtyp rho, const numtyp e, const numtyp cv, numtyp pc[2])
{
  numtyp T = e/cv;
  numtyp beta = ucl_recip(T); // (numtyp)1.0 / T;
  numtyp beta_sqrt = ucl_sqrt(beta);
  numtyp x = rho * ucl_sqrt(beta_sqrt);

  numtyp xsq = x * x;
  numtyp xpow3 = xsq * x;
  numtyp xpow4 = xsq * xsq;

  /* differential of Helmholtz free energy w.r.t. x */
  numtyp diff_A_NkT = (numtyp)3.629 + (numtyp)7.264*x -
    beta*((numtyp)3.492 - (numtyp)18.698*x + (numtyp)35.505*xsq - (numtyp)31.816*xpow3 +
    (numtyp)11.195*xpow4) - beta_sqrt*((numtyp)5.369 + (numtyp)13.16*x +
    (numtyp)18.525*xsq - (numtyp)17.076*xpow3 + (numtyp)9.32*xpow4) +
    (numtyp)10.4925*xsq + (numtyp)11.46*xpow3 + (numtyp)2.176*xpow4*xpow4*x;

  /* differential of Helmholtz free energy w.r.t. x^2 */
  numtyp d2A_dx2 = (numtyp)7.264 + (numtyp)20.985*x +
     beta*((numtyp)18.698 - (numtyp)71.01*x + (numtyp)95.448*xsq - (numtyp)44.78*xpow3) -
     beta_sqrt*((numtyp)13.16 + (numtyp)37.05*x - (numtyp)51.228*xsq + (numtyp)37.28*xpow3) +
     (numtyp)34.38*xsq + (numtyp)19.584*xpow4*xpow4;

  // p = rho k T * (1 + rho * d(A/(NkT))/drho)
  // dx/drho = rho/x
  pc[0] = rho * T * ((numtyp)1.0 + diff_A_NkT * x); // pressure
  numtyp csq = T * ((numtyp)1.0 + (numtyp)2.0 * diff_A_NkT * x + d2A_dx2 * x * x); // soundspeed squared
  if (csq > (numtyp)0.0) {
    pc[1] = ucl_sqrt(csq); // soundspeed
  } else {
    pc[1] = (numtyp)0.0;
  }
}


__kernel void k_sph_lj(const __global numtyp4 *restrict x_,
                       const __global numtyp4 *restrict extra,
                       const __global numtyp4 *restrict coeff,
                       const __global numtyp *restrict mass,
                       const int lj_types,
                       const __global numtyp *restrict sp_lj,
                       const __global int * dev_nbor,
                       const __global int * dev_packed,
                       __global acctyp3 *restrict ans,
                       __global acctyp *restrict engv,
                       __global acctyp *restrict drhoE,
                       const int eflag, const int vflag,
                       const int inum, const int nbor_pitch,
                       const __global numtyp4 *restrict v_,
                       const int dimension, const int t_per_atom) {
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }
  acctyp2 drhoEacc;
  drhoEacc.x = drhoEacc.y = (acctyp)0;

  if (ii<inum) {
    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp mass_itype = mass[itype];
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];

    const numtyp4 extrai = extra[i];
    numtyp rhoi = extrai.x;
    numtyp esphi = extrai.y;
    numtyp cvi = extrai.z;

    // compute pressure of particle i with LJ EOS
    numtyp fci[2];
    LJEOS2(rhoi, esphi, cvi, fci);
    numtyp fi = fci[0];
    numtyp ci = fci[1];
    fi /= (rhoi * rhoi);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<coeff[mtype].z) { // cutsq[itype][jtype]
        numtyp mass_jtype = mass[jtype];
        const numtyp coeffx=coeff[mtype].x;  // viscosity[itype][jtype]
        const numtyp coeffy=coeff[mtype].y;  // cut[itype][jtype]

        const numtyp4 extraj = extra[j];
        numtyp rhoj = extraj.x;
        numtyp esphj = extraj.y;
        numtyp cvj = extraj.z;

        numtyp h = coeffy; // cut[itype][jtype]
        numtyp ih = ucl_recip(h); // (numtyp)1.0 / h;
        numtyp ihsq = ih * ih;
        numtyp ihcub = ihsq * ih;

        numtyp wfd = h - ucl_sqrt(rsq);
        if (dimension == 3) {
          // Lucy Kernel, 3d
          wfd = (numtyp)-25.066903536973515383 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
          // Lucy Kernel, 2d
          wfd = (numtyp)-19.098593171027440292 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        // function call to LJ EOS
        numtyp fcj[2];
        LJEOS2(rhoj, esphj, cvj, fcj);
        numtyp fj = fcj[0];
        numtyp cj = fcj[1];
        fj /= (rhoj * rhoj);

        // apply long-range correction to model a LJ fluid with cutoff
        // this implies that the modelled LJ fluid has cutoff == SPH cutoff
        numtyp lrc = (numtyp)-11.1701 * (ihcub * ihcub * ihcub - (numtyp)1.5 * ihcub);
        fi += lrc;
        fj += lrc;

        // dot product of velocity delta and distance vector
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp delVdotDelR = delx*delvx + dely*delvy + delz*delvz;

        // artificial viscosity (Monaghan 1992)
        numtyp fvisc = (numtyp)0;
        if (delVdotDelR < (numtyp)0) {
          numtyp mu = h * delVdotDelR / (rsq + (numtyp)0.01 * h * h);
          fvisc = -coeffx * (ci + cj) * mu / (rhoi + rhoj); // viscosity[itype][jtype]
        }

        // total pair force & thermal energy increment
        numtyp force = -mass_itype * mass_jtype * (fi + fj + fvisc) * wfd;
        numtyp deltaE = (numtyp)-0.5 * force * delVdotDelR;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // and change in density, drho[i]
        drhoEacc.x += mass_jtype * delVdotDelR * wfd;

        // change in thermal energy, desph[i]
        drhoEacc.y += deltaE;

        if (EVFLAG && eflag) {
          numtyp e = (numtyp)0;
          energy+=e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
  store_drhoE(drhoEacc,ii,inum,tid,t_per_atom,offset,i,drhoE);
}

__kernel void k_sph_lj_fast(const __global numtyp4 *restrict x_,
                            const __global numtyp4 *restrict extra,
                            const __global numtyp4 *restrict coeff_in,
                            const __global numtyp *restrict mass,
                            const __global numtyp *restrict sp_lj_in,
                            const __global int * dev_nbor,
                            const __global int * dev_packed,
                            __global acctyp3 *restrict ans,
                            __global acctyp *restrict engv,
                            __global acctyp *restrict drhoE,
                            const int eflag, const int vflag,
                            const int inum, const int nbor_pitch,
                            const __global numtyp4 *restrict v_,
                            const int dimension, const int t_per_atom) {
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
  }
  __syncthreads();
  #else
  const numtyp coeffx=coeff_in[ONETYPE].x;   // viscosity[itype][jtype]
  const numtyp coeffy=coeff_in[ONETYPE].y;   // cut[itype][jtype]
  const numtyp cutsq_p=coeff_in[ONETYPE].z;  // cutsq[itype][jtype]
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }
  acctyp2 drhoEacc;
  drhoEacc.x = drhoEacc.y = (acctyp)0;

  if (ii<inum) {
    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    numtyp mass_itype = mass[iw];
    #ifndef ONETYPE
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    #endif
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];

    const numtyp4 extrai = extra[i];
    numtyp rhoi = extrai.x;
    numtyp esphi = extrai.y;
    numtyp cvi = extrai.z;

    // compute pressure of particle i with LJ EOS
    numtyp fci[2];
    LJEOS2(rhoi, esphi, cvi, fci);
    numtyp fi = fci[0];
    numtyp ci = fci[1];
    fi /= (rhoi * rhoi);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      #ifndef ONETYPE
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      const numtyp cutsq_p=coeff[mtype].z; // cutsq[itype][jtype];
      #endif
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq_p) {
        numtyp mass_jtype = mass[jtype];
        #ifndef ONETYPE
        const numtyp coeffx=coeff[mtype].x;  // viscosity[itype][jtype]
        const numtyp coeffy=coeff[mtype].y;  // cut[itype][jtype]
        #endif
        const numtyp4 extraj = extra[j];
        numtyp rhoj = extraj.x;
        numtyp esphj = extraj.y;
        numtyp cvj = extraj.z;

        numtyp h = coeffy; // cut[itype][jtype]
        numtyp ih = ucl_recip(h); // (numtyp)1.0 / h;
        numtyp ihsq = ih * ih;
        numtyp ihcub = ihsq * ih;

        numtyp wfd = h - ucl_sqrt(rsq);
        if (dimension == 3) {
          // Lucy Kernel, 3d
          wfd = (numtyp)-25.066903536973515383 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
          // Lucy Kernel, 2d
          wfd = (numtyp)-19.098593171027440292 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        // function call to LJ EOS
        numtyp fcj[2];
        LJEOS2(rhoj, esphj, cvj, fcj);
        numtyp fj = fcj[0];
        numtyp cj = fcj[1];
        fj /= (rhoj * rhoj);

        // apply long-range correction to model a LJ fluid with cutoff
        // this implies that the modelled LJ fluid has cutoff == SPH cutoff
        numtyp lrc = (numtyp)-11.1701 * (ihcub * ihcub * ihcub - (numtyp)1.5 * ihcub);
        fi += lrc;
        fj += lrc;

        // dot product of velocity delta and distance vector
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp delVdotDelR = delx*delvx + dely*delvy + delz*delvz;

        // artificial viscosity (Monaghan 1992)
        numtyp fvisc = (numtyp)0;
        if (delVdotDelR < (numtyp)0) {
          numtyp mu = h * delVdotDelR / (rsq + (numtyp)0.01 * h * h);
          fvisc = -coeffx * (ci + cj) * mu / (rhoi + rhoj); // viscosity[itype][jtype]
        }

        // total pair force & thermal energy increment
        numtyp force = -mass_itype * mass_jtype * (fi + fj + fvisc) * wfd;
        numtyp deltaE = (numtyp)-0.5 * force * delVdotDelR;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // and change in density, drho[i]
        drhoEacc.x += mass_jtype * delVdotDelR * wfd;

        // change in thermal energy, desph[i]
        drhoEacc.y += deltaE;

        if (EVFLAG && eflag) {
          numtyp e = (numtyp)0;
          energy+=e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii

  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag, ans,engv);
  store_drhoE(drhoEacc,ii,inum,tid,t_per_atom,offset,i,drhoE);
}

