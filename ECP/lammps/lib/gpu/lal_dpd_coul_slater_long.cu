#include "hip/hip_runtime.h"
// **************************************************************************
//                                   dpd.cu
//                             -------------------
//                           Eddy BARRAUD (IFPEN/Sorbonne)
//                           Trung Dac Nguyen (U Chicago)
//
//  Device code for acceleration of the dpd/coul/slater/long pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : May 28, 2024
//    email                : eddy.barraud@outlook.fr
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( vel_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture_2d( vel_tex,int4);
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#define EPSILON (numtyp)1.0e-10

//#define _USE_UNIFORM_SARU_LCG
//#define _USE_UNIFORM_SARU_TEA8
//#define _USE_GAUSSIAN_SARU_LCG

#if !defined(_USE_UNIFORM_SARU_LCG) && !defined(_USE_UNIFORM_SARU_TEA8) && !defined(_USE_GAUSSIAN_SARU_LCG)
#define _USE_UNIFORM_SARU_LCG
#endif

// References:
// 1. Y. Afshar, F. Schmid, A. Pishevar, S. Worley, Comput. Phys. Comm. 184 (2013), 1119–1128.
// 2. C. L. Phillips, J. A. Anderson, S. C. Glotzer, Comput. Phys. Comm. 230 (2011), 7191-7201.
// PRNG period = 3666320093*2^32 ~ 2^64 ~ 10^19

#define LCGA 0x4beb5d59 /* Full period 32 bit LCG */
#define LCGC 0x2600e1f7
#define oWeylPeriod 0xda879add /* Prime period 3666320093 */
#define oWeylOffset 0x8009d14b
#define TWO_N32 0.232830643653869628906250e-9f /* 2^-32 */

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0]
// using the inherent LCG, then multiply u with sqrt(3) to "match"
// with a normal random distribution.
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
// Curly brackets to make variables local to the scope.
#ifdef _USE_UNIFORM_SARU_LCG
#define SQRT3 (numtyp)1.7320508075688772935274463
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  state = LCGA*state + LCGC;                                                  \
  wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod);   \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0] using TEA8
// then multiply u with sqrt(3) to "match" with a normal random distribution
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
#ifdef _USE_UNIFORM_SARU_TEA8
#define SQRT3 (numtyp)1.7320508075688772935274463
#define k0 0xA341316C
#define k1 0xC8013EA4
#define k2 0xAD90777D
#define k3 0x7E95761E
#define delta 0x9e3779b9
#define rounds 8
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int sum = 0;                                                       \
  for (int i=0; i < rounds; i++) {                                            \
    sum += delta;                                                             \
    state += ((wstate<<4) + k0)^(wstate + sum)^((wstate>>5) + k1);            \
    wstate += ((state<<4) + k2)^(state + sum)^((state>>5) + k3);              \
  }                                                                           \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns two uniformly distributed random numbers r1 and r2 in [-1.0;1.0],
// and uses the polar method (Marsaglia's) to transform to a normal random value
// This is used to compared with CPU DPD using RandMars::gaussian()
#ifdef _USE_GAUSSIAN_SARU_LCG
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state=0x12345678;                                              \
  unsigned int wstate=12345678;                                               \
  state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));                      \
  wstate = (state + seed2) ^ (((signed int)state)>>8);                        \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int v, s;                                                          \
  numtyp r1, r2, rsq;                                                         \
  while (1) {                                                                 \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r1 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r2 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    rsq = r1 * r1 + r2 * r2;                                                  \
    if (rsq < (numtyp)1.0) break;                                             \
  }                                                                           \
  numtyp fac = ucl_sqrt((numtyp)-2.0*log(rsq)/rsq);                           \
  randnum = r2*fac;                                                           \
}
#endif

__kernel void k_dpd_coul_slater_long(const __global numtyp4 *restrict x_,
                    const __global numtyp4 *restrict extra,
                    const __global numtyp4 *restrict coeff,
                    const int lj_types,
                    const __global numtyp *restrict sp_lj,
                    const __global numtyp *restrict sp_cl_in,
                    const __global numtyp *restrict sp_sqrt,
                    const __global int * dev_nbor,
                    const __global int * dev_packed,
                    __global acctyp3 *restrict ans,
                    __global acctyp *restrict engv,
                    const int eflag, const int vflag, const int inum,
                    const int nbor_pitch,
                    const __global numtyp4 *restrict v_,
                    const __global numtyp4 *restrict cutsq,
                    const numtyp dtinvsqrt, const int seed,
                    const int timestep, const numtyp qqrd2e, 
                    const numtyp g_ewald, const numtyp lamda,
                    const int tstat_only,
                    const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_cl[4];
  sp_cl[0]=sp_cl_in[0];
  sp_cl[1]=sp_cl_in[1];
  sp_cl[2]=sp_cl_in[2];
  sp_cl[3]=sp_cl_in[3];

  int n_stride;
  local_allocate_store_charge();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp e_coul, energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    numtyp qtmp = extra[i].x; // q[i]
    numtyp lamdainv = ucl_recip(lamda);

    numtyp factor_dpd, factor_sqrt;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      factor_dpd = sp_lj[sbmask(j)];
      factor_sqrt = sp_sqrt[sbmask(j)];
      numtyp factor_coul;
      factor_coul = (numtyp)1.0-sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;

      // cutsq[mtype].x -> global squared cutoff
      if (rsq<cutsq[mtype].x) {
        numtyp r=ucl_sqrt(rsq);
        numtyp force_dpd = (numtyp)0.0;
        numtyp force_coul = (numtyp)0.0;

        // apply DPD force if distance below DPD cutoff
        // cutsq[mtype].y -> DPD squared cutoff
        if (rsq < cutsq[mtype].y && r > EPSILON) {

          numtyp rinv=ucl_recip(r);
          numtyp delvx = iv.x - jv.x;
          numtyp delvy = iv.y - jv.y;
          numtyp delvz = iv.z - jv.z;
          numtyp dot = delx*delvx + dely*delvy + delz*delvz;
          numtyp wd = (numtyp)1.0 - r/coeff[mtype].w;

          unsigned int tag1=itag, tag2=jtag;
          if (tag1 > tag2) {
            tag1 = jtag; tag2 = itag;
          }

          numtyp randnum = (numtyp)0.0;
          saru(tag1, tag2, seed, timestep, randnum);

          // conservative force = a0 * wd, or 0 if tstat only
          // drag force = -gamma * wd^2 * (delx dot delv) / r
          // random force = sigma * wd * rnd * dtinvsqrt;

          if (!tstat_only) force_dpd = coeff[mtype].x*wd;
          force_dpd -= coeff[mtype].y*wd*wd*dot*rinv;
          force_dpd *= factor_dpd;
          force_dpd += factor_sqrt*coeff[mtype].z*wd*randnum*dtinvsqrt;
          force_dpd *=rinv;

          if (EVFLAG && eflag) {
            // unshifted eng of conservative term:
            // evdwl = -a0[itype][jtype]*r * (1.0-0.5*r/cut[itype][jtype]);
            // eng shifted to 0.0 at cutoff
            numtyp e = (numtyp)0.5*coeff[mtype].x*coeff[mtype].w * wd*wd;
            energy += factor_dpd*e;
          }

        }// if cut_dpdsq
      
        // apply Slater electrostatic force if distance below Slater cutoff 
        // and the two species have a slater coeff
        // cutsq[mtype].z -> Coulombic squared cutoff
        if ( cutsq[mtype].z != 0.0 && rsq < cutsq[mtype].z){
          numtyp r2inv=ucl_recip(rsq);
          numtyp _erfc;
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
          _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;
          numtyp prefactor = extra[j].x;
          prefactor *= qqrd2e * cutsq[mtype].z * qtmp/r;
          numtyp rlamdainv = r * lamdainv;
          numtyp exprlmdainv = ucl_exp((numtyp)-2.0*rlamdainv);
          numtyp slater_term = exprlmdainv*((numtyp)1.0 + ((numtyp)2.0*rlamdainv*((numtyp)1.0+rlamdainv)));
          force_coul = prefactor*(_erfc + EWALD_F*grij*expm2-slater_term);
          if (factor_coul > (numtyp)0) force_coul -= factor_coul*prefactor*((numtyp)1.0-slater_term);
          force_coul *= r2inv;

          if (EVFLAG && eflag) {
            numtyp e_slater = ((numtyp)1.0 + rlamdainv)*exprlmdainv;
            numtyp e = prefactor*(_erfc-e_slater);
            if (factor_coul > (numtyp)0) e -= factor_coul*prefactor*((numtyp)1.0 - e_slater);
            e_coul += e;
          }
        } // if cut_coulsq

        numtyp force = force_coul + force_dpd;
        f.x += delx*force;
        f.y += dely*force;
        f.z += delz*force;

        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      
      } // if cutsq

    } // for nbor
  } // if ii
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,ans,engv);
}

__kernel void k_dpd_coul_slater_long_fast(const __global numtyp4 *restrict x_,
                         const __global numtyp4 *restrict extra,
                         const __global numtyp4 *restrict coeff_in,
                         const __global numtyp *restrict sp_lj_in,
                         const __global numtyp *restrict sp_cl_in,
                         const __global numtyp *restrict sp_sqrt_in,
                         const __global int * dev_nbor,
                         const __global int * dev_packed,
                         __global acctyp3 *restrict ans,
                         __global acctyp *restrict engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch,
                         const __global numtyp4 *restrict v_,
                         const __global numtyp4 *restrict cutsq_in,
                         const numtyp dtinvsqrt, const int seed,
                         const int timestep, const numtyp qqrd2e, 
                         const numtyp g_ewald, const numtyp lamda,
                         const int tstat_only,
                         const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 cutsq[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  __local numtyp sp_sqrt[4];
  /// COUL Init
  __local numtyp sp_cl[4];
  if (tid<4) {
    sp_lj[tid]=sp_lj_in[tid];
    sp_sqrt[tid]=sp_sqrt_in[tid];
    sp_cl[tid]=sp_cl_in[tid];
  }
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
    cutsq[tid]=cutsq_in[tid];
  }

  __syncthreads();
  

  int n_stride;
  local_allocate_store_charge();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp e_coul, energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    numtyp qtmp = extra[i].x; // q[i]
    numtyp lamdainv = ucl_recip(lamda);

    numtyp factor_dpd, factor_sqrt;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      factor_dpd = sp_lj[sbmask(j)];
      factor_sqrt = sp_sqrt[sbmask(j)];
      numtyp factor_coul;
      factor_coul = (numtyp)1.0-sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype+jx.w;
      
      /// cutsq.x = cutsq, cutsq.y = cut_dpdsq, cutsq.z = cut_slatersq
      if (rsq<cutsq[mtype].x) {
        numtyp r=ucl_sqrt(rsq);
        numtyp force_dpd = (numtyp)0.0;
        numtyp force_coul = (numtyp)0.0;

        // apply DPD force if distance below DPD cutoff
        // cutsq[mtype].y -> DPD squared cutoff
        if (rsq < cutsq[mtype].y && r > EPSILON) {

          numtyp rinv=ucl_recip(r);
          numtyp delvx = iv.x - jv.x;
          numtyp delvy = iv.y - jv.y;
          numtyp delvz = iv.z - jv.z;
          numtyp dot = delx*delvx + dely*delvy + delz*delvz;
          numtyp wd = (numtyp)1.0 - r/coeff[mtype].w;

          unsigned int tag1=itag, tag2=jtag;
          if (tag1 > tag2) {
            tag1 = jtag; tag2 = itag;
          }

          numtyp randnum = (numtyp)0.0;
          saru(tag1, tag2, seed, timestep, randnum);

          // conservative force = a0 * wd, or 0 if tstat only
          // drag force = -gamma * wd^2 * (delx dot delv) / r
          // random force = sigma * wd * rnd * dtinvsqrt;
          /// coeff.x = a0, coeff.y = gamma, coeff.z = sigma, coeff.w = cut_dpd

          if (!tstat_only) force_dpd = coeff[mtype].x*wd;
          force_dpd -= coeff[mtype].y*wd*wd*dot*rinv;
          force_dpd *= factor_dpd;
          force_dpd += factor_sqrt*coeff[mtype].z*wd*randnum*dtinvsqrt;
          force_dpd *=rinv;

          if (EVFLAG && eflag) {
            // unshifted eng of conservative term:
            // evdwl = -a0[itype][jtype]*r * (1.0-0.5*r/cut[itype][jtype]);
            // eng shifted to 0.0 at cutoff
            numtyp e = (numtyp)0.5*coeff[mtype].x*coeff[mtype].w * wd*wd;
            energy += factor_dpd*e;
          }

        }// if cut_dpdsq
      
        // apply Slater electrostatic force if distance below Slater cutoff 
        // and the two species have a slater coeff
        // cutsq[mtype].z -> Coulombic squared cutoff
        if ( cutsq[mtype].z != 0.0 && rsq < cutsq[mtype].z){
          numtyp r2inv=ucl_recip(rsq);
          numtyp _erfc;
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
          _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;
          numtyp prefactor = extra[j].x;
          prefactor *= qqrd2e * cutsq[mtype].z * qtmp/r;
          numtyp rlamdainv = r * lamdainv;
          numtyp exprlmdainv = ucl_exp((numtyp)-2.0*rlamdainv);
          numtyp slater_term = exprlmdainv*((numtyp)1.0 + ((numtyp)2.0*rlamdainv*((numtyp)1.0+rlamdainv)));
          force_coul = prefactor*(_erfc + EWALD_F*grij*expm2-slater_term);
          if (factor_coul > (numtyp)0) force_coul -= factor_coul*prefactor*((numtyp)1.0-slater_term);
          force_coul *= r2inv;

          if (EVFLAG && eflag) {
            numtyp e_slater = ((numtyp)1.0 + rlamdainv)*exprlmdainv;
            numtyp e_sf = prefactor*(_erfc-e_slater);
            if (factor_coul > (numtyp)0) e_sf -= factor_coul*prefactor*((numtyp)1.0 - e_slater);
            e_coul += e_sf;
          }
        } // if cut_coulsq

        numtyp force = force_coul + force_dpd;
        f.x += delx*force;
        f.y += dely*force;
        f.z += delz*force;

        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      
      } // if cutsq

    } // for nbor
  } // if ii
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,ans,engv);
}

